#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"CU_GIN_WL.h"
#include<omp.h>
#include<hip/hip_runtime.h>

namespace CU_WL {

__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float epsilon,
			int numOfNodes, int numOfDirectedEdges, int numOfFeatures, float* outputFeatureMatrix) {

    int i = threadIdx.x;
    int j= blockIdx.x;
    int k = 150;

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
    //printf("thread_idx is: %d\n", thread_idx);

    if (thread_idx < numOfNodes*numOfFeatures*numOfDirectedEdges) {
    
    // get indices of the thread
    
    printf("thread_idx is: %d\n", thread_idx);

    printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);

    const int64_t id_exEdges = (thread_idx / numOfNodes * numOfFeatures);

    const int64_t id_exNodes = (thread_idx / numOfDirectedEdges * numOfFeatures);
	    
    const int64_t id_exFeatures = (thread_idx / numOfNodes * numOfDirectedEdges);

    // if an incoming edge to respected node
    if( *(edgeIndex + numOfDirectedEdges + id_exEdges) == id_exNodes )
	*(featureTensor + numOfFeatures*( *(edgeIndex + numOfDirectedEdges + id_exEdges) )
	    + id_exFeatures) = *(src + thread_idx);
    }



    // the below part is on hold due to kernel update
    //if(i < numOfNodes) {
                //for(int j=0; j<numOfDirectedEdges; j++) {
			//if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
				//for(int k=0; k<numOfFeatures; k++) {
					//*(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k);
				//}
			//}
                //}

		//for(int k=0; k<numOfFeatures; k++) {
			//*(outputFeatureMatrix + i*numOfFeatures + k) = (1 + epsilon)*(*(outputFeatureMatrix + i*numOfFeatures + k)) + *(aggregationVar + i*numOfFeatures + k);
		//}
    //}
}

} // namespace end
