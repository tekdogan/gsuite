#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

#define DIRECTED_EDGES 4

#define NUM_NODES 3

#define FEATURE_LEN 2





__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, float epsilon, float* featureTensorOutput) {

    int i = threadIdx.x;
    if(i < NUM_NODES) {
                for(int j=0; j<DIRECTED_EDGES; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<FEATURE_LEN; k++) {
                                        aggregationVar[k] += *(featureTensor + FEATURE_LEN*((int)(*(edgeIndex + 1*DIRECTED_EDGES + j))) + j);
                                }
                        }
                }
                *(featureTensorOutput + FEATURE_LEN*i) = aggregationVar[0] + (*(featureTensor + FEATURE_LEN*i))*(1+epsilon);
                *(featureTensorOutput + FEATURE_LEN*i + 1) = aggregationVar[1] + (*(featureTensor + FEATURE_LEN*i + 1))*(1+epsilon);
                aggregationVar[0] = 0.0;
                aggregationVar[1] = 0.0;
        }
}
