#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>


__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, float epsilon, float* featureTensorOutput,
			int numOfNodes, int numOfDirectedEdges, int numOfFeatures) {

    int i = threadIdx.x;
    if(i < numOfNodes) {
                for(int j=0; j<DIRECTED_EDGES; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<numOfFeatures; k++) {
                                        aggregationVar[k] += *(featureTensor + numOfFeatures*((int)(*(edgeIndex + 1*numOfDirectedEdges + j))) + j);
                                }
                        }
                }
                *(featureTensorOutput + numOfFeatures*i) = aggregationVar[0] + (*(featureTensor + numOfFeatures*i))*(1+epsilon);
                *(featureTensorOutput + numOfFeatures*i + 1) = aggregationVar[1] + (*(featureTensor + numOfFeatures*i + 1))*(1+epsilon);
                aggregationVar[0] = 0.0;
                aggregationVar[1] = 0.0;
        }
}
