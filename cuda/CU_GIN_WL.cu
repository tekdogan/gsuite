#include "hip/hip_runtime.h"
#include<iostream>
#include"CU_GIN_WL.h"
#include<hip/hip_runtime.h>

namespace CU_WL {

__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float epsilon,
			int numOfNodes, int numOfDirectedEdges, int numOfFeatures, float* outputFeatureMatrix) {

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
    //printf("thread_idx is: %d\n", thread_idx);

    if (thread_idx < numOfNodes*numOfFeatures*numOfDirectedEdges) {
    
    // get indices of the thread
    
    printf("thread_idx is: %d\n", thread_idx);

    printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);

    const int64_t id_exEdges = (thread_idx / numOfNodes * numOfFeatures);

    const int64_t id_exNodes = (thread_idx / numOfDirectedEdges * numOfFeatures);

    const int64_t id_exFeatures = (thread_idx / numOfNodes * numOfDirectedEdges);

    // if an incoming edge to respected node
    if( *(edgeIndex + numOfDirectedEdges + id_exEdges) == id_exNodes )
	// then apply aggregation scheme of GCN
	// to corresponding node's feature
	*(featureTensor + numOfFeatures*( *(edgeIndex + numOfDirectedEdges + id_exEdges) )
	    + id_exFeatures) = *(src + thread_idx);
    }
	
    //sync threads before output update
    __syncthreads();

    // update output feature values
    outputFeatureMatrix + numOfFeatures*id_exNodes + id_exFeatures =
	    (1 + epsilon)*(*(outputFeatureMatrix + numOfFeatures*id_exNodes + id_exFeatures)) +
	    *(aggregationVar + numOfFeatures*id_exNodes + id_exFeatures);



    // the below part is on hold due to kernel update
    //if(i < numOfNodes) {
                //for(int j=0; j<numOfDirectedEdges; j++) {
			//if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
				//for(int k=0; k<numOfFeatures; k++) {
					//*(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k);
				//}
			//}
                //}

		//for(int k=0; k<numOfFeatures; k++) {
			//*(outputFeatureMatrix + i*numOfFeatures + k) = (1 + epsilon)*(*(outputFeatureMatrix + i*numOfFeatures + k)) + *(aggregationVar + i*numOfFeatures + k);
		//}
    //}
}

} // namespace end
