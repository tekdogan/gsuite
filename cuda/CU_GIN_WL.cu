#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"CU_GIN_WL.h"
#include<omp.h>
#include<hip/hip_runtime.h>

namespace CU_WL {

__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float epsilon, float* featureTensorOutput,
			int numOfNodes, int numOfDirectedEdges, int numOfFeatures, float* outputFeatureMatrix) {

    int i = threadIdx.x;
    if(i < numOfNodes) {
                for(int j=0; j<numOfDirectedEdges; j++) {
			if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
				for(int k=0; k<numOfFeatures; k++) {
					*(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k);
				}
			}
                }

		for(int k=0; k<numOfFeatures; k++) {
			*(outputFeatureMatrix + i*numOfFeatures + k) = (1 + epsilon)*(*(outputFeatureMatrix + i*numOfFeatures + k)) + *(aggregationVar + i*numOfFeatures + k);
		}
    }
}

} // namespace end
