#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"CU_GIN_WL.h"
#include<omp.h>
#include<hip/hip_runtime.h>

namespace CU_WL {

__global__ void GINLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float epsilon,
			int numOfNodes, int numOfDirectedEdges, int numOfFeatures, float* outputFeatureMatrix) {

    int i = threadIdx.x;
    int j= blockIdx.x;
    int k = 150;

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
    //printf("thread_idx is: %d\n", thread_idx);

    if (thread_idx < numOfNodes*numOfFeatures*numOfDirectedEdges) {
    
    // get indices of the thread
    
    printf("thread_idx is: %d\n", thread_idx);

    printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);

    int64_t id_exEdges = (thread_idx / numOfNodes * numOfFeatures);
    
    int64_t index_info = thread_idx % (numOfFeatures*numOfDirectedEdges);
    
    int64_t id_r = (idx / numOfNodes);
    
    int64_t id_c = (id_r / numOfFeatures);

    }




    //if(i < numOfNodes) {
                //for(int j=0; j<numOfDirectedEdges; j++) {
			if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
				//for(int k=0; k<numOfFeatures; k++) {
					*(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k);
				//}
			}
                //}

		//for(int k=0; k<numOfFeatures; k++) {
			*(outputFeatureMatrix + i*numOfFeatures + k) = (1 + epsilon)*(*(outputFeatureMatrix + i*numOfFeatures + k)) + *(aggregationVar + i*numOfFeatures + k);
		//}
    //}
}

} // namespace end
