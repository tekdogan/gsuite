#include<iostream>
#include"CU_GIN_WL.h"
#include<hip/hip_runtime.h>
#include"index_select.h"
#include"linear.h"

#include "scatter_cuda.h"
#include "index_select.h"
#include "linear.h"

namespace CU_WL {


float* GINLayer(int* h_edgeIndex, float* h_featureVector, int numOfNodes, int numOfFeatures,
			int numOfEdges, int outputSize, float eps) {


	int* h_edgeIndexSrc;
	int* h_edgeIndexDst;
	h_edgeIndexSrc = (int*)calloc(numOfEdges, sizeof(int));
        h_edgeIndexDst = (int*)calloc(numOfEdges, sizeof(int));

	memcpy(h_edgeIndexSrc, h_edgeIndex, numOfEdges);
	memcpy(h_edgeIndexDst, h_edgeIndex + numOfEdges, numOfEdges);


        float* indexSelectOutput = index_select(h_featureVector, numOfNodes, numOfFeatures, 0, h_edgeIndexSrc, numOfEdges);


	float* aggrOutput = scatter_cuda(indexSelectOutput, h_edgeIndexDst, 1, "sum", numOfEdges, numOfEdges, numOfFeatures, numOfNodes, numOfFeatures);
	
	for(int i = 0; i<numOfNodes*numOfFeatures; i++) {
		aggrOutput[i] += (1 + eps) * h_featureVector[i];
	}


	float* output = linear(aggrOutput, numOfNodes, numOfFeatures, numOfNodes, outputSize);	

	
	free(h_edgeIndexSrc);
	free(h_edgeIndexDst);
	free(indexSelectOutput);
	free(aggrOutput);	

	return output;
}

} // namespace end
