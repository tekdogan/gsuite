// This function is designed for a temporary use of
// direct usage of CUDA data loader utility from
// a cpp based main file.

//#include"DataLoader.h"
#include "../scatter_cuda.h"
#include "../Data_Util.h"
#include <iostream>

#include <unistd.h>

/*

edge_index 
source 0,0,0,1,1,2,2,2,3,3,4,4
dest   1,2,4,0,3,0,3,4,1,2,0,2
*/

int main(int argc, char *argv[]) {

        //if(argc == 1) {
        //      std::cout << "Please pass a parameter to executable. (e.g. ./cudaDataLoader.o 2)\n";
        //}

        //LoadData(atoi(argv[1]));

	/*
		3x4 node to feature vector
	*/
        float h_src[48] = 
	{
		    0,0,0,0,
		    0,0,0,0,
		    0,0,0,0,
                    1,1,1,1,
		    1,1,1,1,
                    2,2,2,2,
                    2,2,2,2,
		    2,2,2,2,
                    3,3,3,3,
		    3,3,3,3,
		    4,4,4,4,
		    4,4,4,4
	};
	
	int featureLen = 4;
	int nodeCount = 5;
	int edgeCount = 12;	

	float h_edgeSource[12] = {0,0,0,1,1,2,2,2,3,3,4,4};
	float h_edgeDest[12] = {1,2,4,0,3,0,3,4,1,2,0,2};

        int srcRows = 5;
        int srcCols = 4;

        int dstSize = nodeCount * featureLen;


	float* d_src;
	float* d_index;


	hipMalloc((void**) &d_src, (featureLen * edgeCount)*sizeof(float));
	hipMemcpy(d_src, h_src, (featureLen * edgeCount)*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**) &d_index, (edgeCount)*sizeof(float));
        hipMemcpy(d_index, h_edgeDest, (edgeCount)*sizeof(float), hipMemcpyHostToDevice);

	float* a = scatter_cuda(d_src, d_index, 1, "sum", nodeCount, featureLen, edgeCount);

	sleep(10);

	
        float *h_out = (float*)calloc(nodeCount*featureLen, sizeof(float));

	hipMemcpy(h_out, a, (nodeCount*featureLen*sizeof(float)), hipMemcpyDeviceToHost);

	printDenseMatrix(a, nodeCount, featureLen);

        return 0;
}


