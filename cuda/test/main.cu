// This function is designed for a temporary use of
// direct usage of CUDA data loader utility from
// a cpp based main file.

//#include"DataLoader.h"
#include "index_select.h"
#include "Data_Util.h"
#include <iostream>

int main(int argc, char *argv[]) {

	//if(argc == 1) {
	//	std::cout << "Please pass a parameter to executable. (e.g. ./cudaDataLoader.o 2)\n";
	//}

	//LoadData(atoi(argv[1]));

	float h_src[9] = {1,2,3,
		    4,5,6,
		    7,8,9};

	int srcRows = 3;
	int srcCols = 3;

	int dim = 0;

	int h_indices[2] = {0,2};

	int indSize = 2;

	int dstSize = 6;

	float *h_out = (float*)calloc(dstSize, sizeof(float));

	float *d_src, *d_out;
	int *d_indices;

	hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
	hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**) &d_out, dstSize*sizeof(float));
        hipMemcpy(d_out, h_out, dstSize*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**) &d_indices, indSize*sizeof(int));
	hipMemcpy(d_indices, h_indices, indSize*sizeof(int), hipMemcpyHostToDevice);

	index_select(h_src, srcRows, srcCols,
                    dim, h_indices, indSize, h_out);

	//hipMemcpy(h_out, d_out, dstSize*sizeof(float), hipMemcpyDeviceToHost);

	printDenseMatrix(h_out, dstSize/srcCols, srcCols);

	return 0;
}
