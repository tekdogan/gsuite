// This function is designed for a temporary use of
// direct usage of CUDA data loader utility from
// a cpp based main file.

//#include"DataLoader.h"
#include "linear.h"
#include "Data_Util.h"
#include <iostream>

int main(int argc, char *argv[]) {

	//if(argc == 1) {
	//	std::cout << "Please pass a parameter to executable. (e.g. ./cudaDataLoader.o 2)\n";
	//}

	//LoadData(atoi(argv[1]));

	float h_src[9] = {1,2,3,
		          4,5,6,
		          7,8,9};

	int srcRows = 3;
	int srcCols = 3;

	int outRows = srcRows;
	int outCols = 3;

	float *h_out = (float*)calloc(outRows*outCols, sizeof(float));

	float *d_src, *d_out;

	//hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
	//hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

	//hipMalloc((void**) &d_out, dstSize*sizeof(float));
        //hipMemcpy(d_out, h_out, dstSize*sizeof(float), hipMemcpyHostToDevice);

	//hipMalloc((void**) &d_indices, indSize*sizeof(int));
	//hipMemcpy(d_indices, h_indices, indSize*sizeof(int), hipMemcpyHostToDevice);

	linear(h_src, srcRows, srcCols,
              h_out, outRows, outCols);

	//hipMemcpy(h_out, d_out, dstSize*sizeof(float), hipMemcpyDeviceToHost);

	printDenseMatrix(h_src, outRows, outCols);

	return 0;
}
