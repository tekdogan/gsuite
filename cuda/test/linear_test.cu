// This function is designed for a temporary use of
// direct usage of CUDA data loader utility from
// a cpp based main file.

//#include"DataLoader.h"
#include "linear.h"
#include "Data_Util.h"
#include <iostream>

int main(int argc, char *argv[]) {

	//if(argc == 1) {
	//	std::cout << "Please pass a parameter to executable. (e.g. ./cudaDataLoader.o 2)\n";
	//}

	//LoadData(atoi(argv[1]));

	float h_src[6] = {1,2,
		          3,4,
		          5,6};

	int srcRows = 3;
	int srcCols = 2;

	int outRows = srcRows;
	int outCols = 1;

	float *h_out = (float*)calloc(outRows*outCols, sizeof(float));

	float *d_src, *d_out;

	//hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
	//hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

	//hipMalloc((void**) &d_out, dstSize*sizeof(float));
        //hipMemcpy(d_out, h_out, dstSize*sizeof(float), hipMemcpyHostToDevice);

	//hipMalloc((void**) &d_indices, indSize*sizeof(int));
	//hipMemcpy(d_indices, h_indices, indSize*sizeof(int), hipMemcpyHostToDevice);

	linear(h_src, srcRows, srcCols,
              h_out, outRows, outCols);

	//hipMemcpy(h_out, d_out, dstSize*sizeof(float), hipMemcpyDeviceToHost);

	//printDenseMatrix(h_out, dstSize/srcCols, srcCols);

	return 0;
}
