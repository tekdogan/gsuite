
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SpMM_GCN.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"


namespace CU_SpMM {


void GCNLayer(float* adjMatrix, float* featureTensor, int n_nodes, int n_edges, int n_features, float* output) {

//    int i = threadIdx.x;
//    if(i < n_nodes) {
        

	// create identity matrix I here
	float *d_I;
	hipMalloc(&d_I,n_nodes * n_nodes * sizeof(float));
	initIdentityGPU<<<16,1024>>>(d_I, n_nodes, n_nodes);
        
        // ----- calculation of (1+e)*I ----- //
	//for(int i=0; i<n_nodes; i++) {

	//}

	//printf("A matrix:\n");
	//printDenseMatrix(adjMatrix, n_nodes, n_nodes);

	// define device matrices
        float *d_A, *d_D, *d_DA, *d_DAD, *d_DADX, *d_X;

	// allocate device A matrix
        hipMalloc(&d_A,n_nodes * n_nodes * sizeof(float));
                
	// migrate A matrix to device
	hipMemcpy(d_A,adjMatrix,n_nodes * n_nodes * sizeof(float),hipMemcpyHostToDevice);

	// ----- calculation of A + (1+e)*I ----- //
	float* D = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	gpu_blas_mmul(d_I, d_A, d_AI, n_nodes, n_nodes, n_nodes, false, false);


	// ----- calculation of (A + (1+e)*I) * X ----- //
	hipMalloc(&d_DA,n_nodes * n_nodes * sizeof(float));
	gpu_blas_mmul(d_A, d_D, d_DA, n_nodes, n_nodes, n_nodes, false, false);
	float* DA = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	//gpu_blas_mmul(D, adjMatrix, DA, n_nodes, n_nodes, n_nodes);
	hipFree(d_A);

	hipMemcpy(DA,d_DA,n_nodes * n_nodes * sizeof(float),hipMemcpyDeviceToHost);
	//printf("DA matrix:\n");
	//printDenseMatrix(DA, n_nodes, n_nodes);
        
	// ----- calculation of D^-1/2 * A^ * D^-1/2 ----- //
	hipMalloc(&d_DAD,n_nodes * n_nodes * sizeof(float));
	gpu_blas_mmul(d_D, d_DA, d_DAD, n_nodes, n_nodes, n_nodes, false, false);
        hipFree(d_DA);
        hipFree(d_D);

	float* DAD = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	hipMemcpy(DAD,d_DAD,n_nodes * n_nodes * sizeof(float),hipMemcpyDeviceToHost);
	//printf("DAD matrix:\n");
	//printDenseMatrix(DAD, n_nodes, n_nodes);
        
	// migrate node feature values from host to device
	hipMalloc(&d_X, n_nodes * n_features * sizeof(float));
	hipMemcpy(d_X, featureTensor, n_nodes * n_features * sizeof(float), hipMemcpyHostToDevice);

	//printf("X matrix:\n");
	//printDenseMatrix(featureTensor, n_nodes, n_features);

	// ----- calculation of D^-1/2 * A^ * D^-1/2 * X ----- //
	hipMalloc(&d_DADX,n_nodes * n_features * sizeof(float));
	gpu_blas_mmul(d_X, d_DAD, d_DADX, n_features, n_nodes, n_features, false, false);
	hipFree(d_DAD);
	hipFree(d_X);

	// copy the result to output
	float* DADX = (float*)calloc(n_nodes * n_features, sizeof(float));
	hipMemcpy(DADX, d_DADX, n_nodes * n_features * sizeof(float), hipMemcpyDeviceToHost);
	memcpy(output, DADX, sizeof(float)*(n_nodes*n_features));

	//printf("DADX matrix:\n");
	//printDenseMatrix(DADX, n_nodes, n_features);
//    }

}

} //namespace end
