
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SpMM_GIN.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"


namespace CU_SpMM {


void GINLayer(float* adjMatrix, float* featureTensor, int n_nodes, int n_edges, int n_features, float* output, float epsilon) {

//    int i = threadIdx.x;
//    if(i < n_nodes) {
        

	// create identity matrix I here
	float *d_I, *d_A, *d_AIX, *d_X;
	hipMalloc(&d_I,n_nodes * n_nodes * sizeof(float));
	initIdentityGPU<<<16,1024>>>(&d_I, n_nodes, n_nodes);
        
        // ----- calculation of (1+e)*I ----- //
	//for(int i=0; i<n_nodes; i++) {

	//}

	printf("I matrix:\n");
	float* I = (float*)calloc(n_nodes * n_nodes, sizeof(float));
	//hipMemcpy(I,d_I,n_nodes * n_nodes * sizeof(float), hipMemcpyDeviceToHost);
	initIdentityMatrix(I, n_nodes, n_nodes);
	printDenseMatrix(I, n_nodes, n_nodes);

	printf("A matrix:\n");
	printDenseMatrix(adjMatrix, n_nodes, n_nodes);

	// allocate device A matrix
        hipMalloc(&d_A,n_nodes * n_nodes * sizeof(float));
                
	// migrate A matrix to device
	hipMemcpy(d_A,adjMatrix,n_nodes * n_nodes * sizeof(float),hipMemcpyHostToDevice);

	// ----- calculation of A + (1+e)*I ----- //
	gpu_blas_mmul(d_I, d_A, d_I, n_nodes, n_nodes, n_nodes, false, false, 1.0, (1.0 + epsilon));
	hipFree(d_A);

	// migrate node feature values from host to device
	hipMalloc(&d_X, n_nodes * n_features * sizeof(float));
	hipMemcpy(d_X, featureTensor, n_nodes * n_features * sizeof(float), hipMemcpyHostToDevice);

	// ----- calculation of (A + (1+e)*I) * X ----- //
	hipMalloc(&d_AIX,n_nodes * n_features * sizeof(float));
	gpu_blas_mmul(d_X, d_I, d_AIX, n_features, n_nodes, n_features, false, false, 1.0, 0.0);
	hipFree(d_I);
	hipFree(d_X);

	// copy the result to output
	hipMemcpy(output, d_AIX, n_nodes * n_features * sizeof(float), hipMemcpyDeviceToHost);
	memcpy(output, d_AIX, sizeof(float)*(n_nodes*n_features));

	//printf("DADX matrix:\n");
	//printDenseMatrix(DADX, n_nodes, n_features);

	hipFree(d_AIX);
//    }

}

} //namespace end
