#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"
#include "linear.h"
#include "index_select.h"


namespace CU_MP {


float* GCNLayer(int* h_edgeIndex, float* h_featureVector,
                int numOfNodes, int numOfFeatures,
                int numOfEdges, int outputSize) {

	// allocations for host variables
	float *h_ones = (float*)calloc(numOfEdges, sizeof(float));
	
	// first part of edgeIndex indicating sources
	int *h_edgeSources = (int*)calloc(numOfEdges, sizeof(int));
	for(int i = 0; i<numOfEdges; i++)
	{
		int d = h_edgeIndex[i];
		h_edgeSources[i] = d;
	}


	memset(h_ones, 1, numOfEdges*sizeof(float));
	
	// compute the node degrees via scatter_add
	float* h_nodeDegrees = scatter_cuda(h_ones, h_edgeSources, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, 1);

	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	
	// linear transform
	float* h_outputLinear = linear(h_featureVector, numOfNodes, numOfFeatures, numOfNodes, outputSize);

	// index select
	float* indexSelectOutput = index_select(h_outputLinear, numOfNodes, outputSize, 0, h_edgeSources, numOfEdges);



	// aggregation via scatter
	int *h_edgeDest = (int*)calloc(numOfEdges, sizeof(int));
	memcpy(h_edgeDest, h_edgeIndex+numOfEdges, numOfEdges*sizeof(int));
	float *output = (float*)calloc(numOfNodes*outputSize, sizeof(float));
	output = scatter_cuda(indexSelectOutput, h_edgeDest, 1, "sum", numOfEdges, numOfEdges, outputSize, numOfNodes, outputSize);
	

	free(h_ones);
	free(h_edgeSources);
	free(h_nodeDegrees);
	free(h_outputLinear);
	free(indexSelectOutput);
	free(h_edgeDest);
	return output;
}


} // namespace end
