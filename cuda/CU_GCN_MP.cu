#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"
#include "linear.h"
#include "index_select.h"


namespace CU_MP {


float* GCNLayer(int* h_edgeIndex, float* h_featureVector, float *h_aggregationVar, float *h_nodeDegrees,
		int numOfNodes, int numOfFeatures, int numOfEdges, int outputSize) {

	// allocations for host variables
	//float *h_nodeDegrees = (float*)calloc(numOfNodes, sizeof(float));
	float *h_ones = (float*)calloc(numOfEdges, sizeof(float));
	
	// first part of edgeIndex indicating sources
	int *h_edgeSources = (int*)calloc(numOfEdges, sizeof(int));
//	memcpy(h_edgeSources, h_edgeIndex, numOfEdges*sizeof(int));
	for(int i = 0; i<numOfEdges; i++)
	{
		int d = h_edgeIndex[i];
		h_edgeSources[i] = d;
	}


	// ones to be used during node degree calculation
	//for(int i=0; i<numOfNodes; i++) {
	//	*(h_ones + i) = 1;
	//}
	memset(h_ones, 1, numOfEdges*sizeof(float));
	
	// compute the node degrees via scatter_add
	h_nodeDegrees = scatter_cuda(h_ones, h_edgeSources, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, 1);

	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	float *h_outputLinear = (float*)calloc(numOfNodes*outputSize, sizeof(float));

	// linear transform
	linear(h_featureVector, numOfNodes, numOfFeatures,
               h_outputLinear, numOfNodes, outputSize);

	// index select
	float *indexSelectOutput = (float*)calloc(numOfEdges*outputSize, sizeof(float));
	indexSelectOutput = index_select(h_outputLinear, numOfNodes, outputSize, 0, h_edgeSources, numOfEdges, indexSelectOutput);

	// aggregation via scatter
	int *h_edgeDest = (int*)calloc(numOfEdges, sizeof(int));
	memcpy(h_edgeDest, h_edgeIndex+numOfEdges, numOfEdges*sizeof(int));
	float *output = (float*)calloc(numOfNodes*outputSize, sizeof(float));
	output = scatter_cuda(indexSelectOutput, h_edgeDest, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, outputSize);
	
	return output;
	
	/*if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfEdges + id_exEdges) )]);
		}
		
	}*/
	
}


__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
