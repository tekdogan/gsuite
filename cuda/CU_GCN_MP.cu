#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"

//#define DIRECTED_EDGES 4

//#define NUM_NODES 3

//#define FEATURE_LEN 3


namespace CU_MP {


void GCNLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees,
		int numOfNodes, int numOfFeatures, int numOfEdges) {

	//int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// compute the node degrees
	auto res = scatter_cuda(nodeDegrees, edgeIndex, 1, "add", numOfNodes, featureSize, edgeIndexSize);
	
	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(numOfNodes + i) = 1/sqrt(numOfNodes);
	}
	
	// aggregation scheme
	auto out = scatter_cuda(featureTensor, edgeIndex, 1, "add", numOfNodes, featureSize, edgeIndexSize);
	
	/*if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfEdges + id_exEdges) )]);
		}
		
	}*/
	
}


__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
