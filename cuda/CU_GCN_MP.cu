#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"


namespace CU_MP {


void GCNLayer(float* h_edgeIndex, float* h_featureVector, float *h_aggregationVar, float *h_nodeDegrees,
		int numOfNodes, int numOfFeatures, int numOfEdges) {

	//int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

	float *d_edgeIndex, *d_featureVector, *d_aggregationVar, *d_nodeDegrees;

	//hipMalloc( (void**) &d_featureVector, numOfNodes*numOfFeatures * sizeof(float));
	//hipMemcpy(d_featureVector, h_featureVector, numOfNodes*numOfFeatures * sizeof(float), hipMemcpyHostToDevice);

	
	// compute the node degrees
	auto res = scatter_cuda(h_nodeDegrees, h_edgeIndex, 1, "sum", numOfNodes, numOfFeatures, numOfEdges);

	//migrate to host	
	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
//		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	// aggregation scheme
	//auto out = scatter_cuda(h_featureVector, h_edgeIndex, 1, "sum", numOfNodes, numOfFeatures, numOfEdges);
	
	/*if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfEdges + id_exEdges) )]);
		}
		
	}*/
	
}


__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
