#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"
#include "linear.h"
#include "index_select.h"


namespace CU_MP {


float* GCNLayer(int* h_edgeIndex, float* h_featureVector,
                int numOfNodes, int numOfFeatures,
                int numOfEdges, int outputSize) {

	// allocations for host variables
	float *h_ones = (float*)calloc(numOfEdges, sizeof(float));
	
	// first part of edgeIndex indicating sources
	int *h_edgeSources = (int*)calloc(numOfEdges, sizeof(int));
	for(int i = 0; i<numOfEdges; i++)
	{
		int d = h_edgeIndex[i];
		h_edgeSources[i] = d;
	}


	memset(h_ones, 1, numOfEdges*sizeof(float));
	
	// compute the node degrees via scatter_add
	float* h_nodeDegrees = scatter_cuda(h_ones, h_edgeSources, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, 1);

	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	float *h_outputLinear = (float*)calloc(numOfNodes*outputSize, sizeof(float));

	// linear transform
	linear(h_featureVector, numOfNodes, numOfFeatures,
               h_outputLinear, numOfNodes, outputSize);

	int *h_edgeIndexSources = (int*)calloc(numOfEdges, sizeof(int));

	// index select
	float *indexSelectOutput = (float*)calloc(numOfEdges*outputSize, sizeof(float));
	indexSelectOutput = index_select(h_outputLinear, numOfNodes, outputSize, 0, h_edgeIndexSources, numOfEdges, indexSelectOutput);



	// aggregation via scatter
	int *h_edgeDest = (int*)calloc(numOfEdges, sizeof(int));
	memcpy(h_edgeDest, h_edgeIndex+numOfEdges, numOfEdges*sizeof(int));
	float *output = (float*)calloc(numOfNodes*outputSize, sizeof(float));
	output = scatter_cuda(indexSelectOutput, h_edgeDest, 1, "sum", numOfEdges, numOfEdges, outputSize, numOfNodes, outputSize);
	

	free(h_ones);
	free(h_edgeSources);
	free(h_outputLinear);
	free(h_edgeIndexSources);
	free(indexSelectOutput);
	free(h_edgeDest);
	return output;
}


} // namespace end
