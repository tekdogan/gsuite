#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"
#include "linear.h"
#include "index_select.h"


namespace CU_MP {


void GCNLayer(int* edgeIndex, float* featureVector, float *aggregationVar, float *nodeDegrees,
		int numOfNodes, int numOfFeatures, int numOfEdges, int outputSize) {

	

	// allocations for host variables
	float *h_edgeIndex = (float*)calloc(numOfEdges * 2, sizeof(float));
	float *h_featureVector = (float*)calloc(numOfNodes * numOfFeatures, sizeof(float));
	float *h_aggregationVar = (float*)calloc(numOfNodes * numOfFeatures, sizeof(float));
	float *h_nodeDegrees = (float*)calloc(numOfNodes, sizeof(float));
	float *h_ones = (float*)calloc(numOfNodes, sizeof(float));
	
	// first part of edgeIndex indicating sources
	int *h_edgeSources = (int*)calloc(numOfEdges, sizeof(int));
	memcpy(h_edgeSources, edgeIndex, numOfEdges*sizeof(int));

	// ones to be used during node degree calculation
	//for(int i=0; i<numOfNodes; i++) {
	//	*(h_ones + i) = 1;
	//}
	memset(h_ones, 1, numOfNodes*sizeof(float));
	
	// compute the node degrees via scatter_add
	h_nodeDegrees = scatter_cuda(h_nodeDegrees, h_edgeSources, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, 1);

	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	float *h_outputLinear = (float*)calloc(numOfNodes*outputSize, sizeof(float));

	// linear transform
	linear(featureVector, numOfNodes, numOfFeatures,
               h_outputLinear, numOfNodes, outputSize);

	int *edgeIndexSources = (int*)calloc(numOfEdges, sizeof(int));

	float *indexSelectOutput = (float*)calloc((numOfEdges)*outputSize, sizeof(float));
	indexSelectOutput = index_select(h_outputLinear, numOfNodes, outputSize, 0, edgeIndexSources, numOfEdges, indexSelectOutput);

	int *h_edgeDest = (int*)calloc(numOfEdges, sizeof(int));
	memcpy(h_edgeDest, edgeIndex+(numOfEdges), numOfEdges);
	float *output = scatter_cuda(indexSelectOutput, h_edgeDest, 1, "sum", numOfEdges, numOfEdges, 1, numOfEdges, outputSize);

	// aggregation scheme
	//auto out = scatter_cuda(h_featureVector, h_edgeIndex, 1, "sum", numOfNodes, numOfFeatures, numOfEdges);
	
	/*if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfEdges + id_exEdges) )]);
		}
		
	}*/
	
}


__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
