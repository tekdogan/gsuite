#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>
#include "scatter_cuda.h"


namespace CU_MP {


void GCNLayer(float* d_edgeIndex, float* d_featureVector, float *d_aggregationVar, float *d_nodeDegrees,
		int numOfNodes, int numOfFeatures, int numOfEdges) {

	
	// variable declarations for host variables
	float *h_edgeIndex, *h_featureVector, *h_aggregationVar, *h_nodeDegrees;

	// allocations for host variables
	*h_edgeIndex = calloc(numOfEdges * 2, sizeof(float));
	*h_featureVector = calloc(numOfNodes * numOfFeatures, sizeof(float));
	*h_aggregationVar = calloc(numOfNodes * numOfFeatures, sizeof(float));
	*h_nodeDegrees = calloc(numOfNodes, sizeof(float));
	
	//hipMalloc( (void**) &d_featureVector, numOfNodes*numOfFeatures * sizeof(float));
	//hipMemcpy(d_featureVector, h_featureVector, numOfNodes*numOfFeatures * sizeof(float), hipMemcpyHostToDevice);

	
	// compute the node degrees via scatter_add
	auto res = scatter_cuda(h_nodeDegrees, h_edgeIndex, 1, "sum", numOfNodes, numOfFeatures, numOfEdges);

	// migrate device degrees output to host
	hipMemcpy(d_nodeDegrees, h_nodeDegrees, numOfNodes * sizeof(float), hipMemcpyHostToDevice);
	
	// sqrt -0.5 of node degrees
	for(int i=0; i<numOfNodes; i++) {
		*(h_nodeDegrees + i) = 1/sqrt(*(h_nodeDegrees + i));
	}
	
	// migrate host degrees back to device
	hipMemcpy(h_nodeDegrees, d_nodeDegrees, numOfNodes * sizeof(float), hipMemcpyDeviceToHost);
	
	// aggregation scheme
	//auto out = scatter_cuda(h_featureVector, h_edgeIndex, 1, "sum", numOfNodes, numOfFeatures, numOfEdges);
	
	/*if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfEdges + id_exEdges) )]);
		}
		
	}*/
	
}


__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
