#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

//#define DIRECTED_EDGES 4

//#define NUM_NODES 3

//#define FEATURE_LEN 3


namespace CU_MP {

void GCNLayer(float* edgeIndex, float* featureTensor, float* aggregationVar, float* nodeDegrees) {

	int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfDirectedEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfDirectedEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfDirectedEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfDirectedEdges + id_exEdges) )]);
			
		}
		
	}
	
/*
        for(int i=0; i<NUM_NODES; i++) {
                for(int j=0; j<DIRECTED_EDGES; j++) {
                        if(edgeIndex[0][j] == i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<FEATURE_LEN; k++) {
                                        aggregationVar[k] += featureTensor[i][k] * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)edgeIndex[1][j]]);
                                }
                        }
                }
                featureTensor[i][0] = aggregationVar[0];
                featureTensor[i][1] = aggregationVar[1];
                aggregationVar[0] = 0.0;
                aggregationVar[1] = 0.0;
        }
*/
}

__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
