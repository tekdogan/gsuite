#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

//#define DIRECTED_EDGES 4

//#define NUM_NODES 3

//#define FEATURE_LEN 3


namespace CU_MP {

void GCNLayer(float* edgeIndex, float* featureTensor, float* aggregationVar, float* nodeDegrees) {
/*
        for(int i=0; i<NUM_NODES; i++) {
                for(int j=0; j<DIRECTED_EDGES; j++) {
                        if(edgeIndex[0][j] == i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<FEATURE_LEN; k++) {
                                        aggregationVar[k] += featureTensor[i][k] * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)edgeIndex[1][j]]);
                                }
                        }
                }
                featureTensor[i][0] = aggregationVar[0];
                featureTensor[i][1] = aggregationVar[1];
                aggregationVar[0] = 0.0;
                aggregationVar[1] = 0.0;
        }
*/
}

__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	if(i < numOfNodes) {
                for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                }
                        }
                }
                *(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                *(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                *(aggregationVar + i*numOfFeatures) = 0.0;
                *(aggregationVar + i*numOfFeatures + 1) = 0.0;
        }


}


} // namespace end
