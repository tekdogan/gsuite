#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"C_GCN_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

//#define DIRECTED_EDGES 4

//#define NUM_NODES 3

//#define FEATURE_LEN 3


namespace CU_MP {

void GCNLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees,
	      	int numOfNodes, int numOfFeatures, int numOfEdges) {

	int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread_idx < numOfNodes*numOfFeatures) {
		
		const int64_t id_exEdges = (thread_idx % numOfDirectedEdges);
		
		const int64_t id_exNodes = (thread_idx / numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfDirectedEdges + id_exEdges) == id_exNodes ) {
			// then apply aggregation scheme of GCN
			// to corresponding node's feature
			*(aggregationVar + (int)numOfFeatures*( (int)*(edgeIndex + numOfDirectedEdges + id_exEdges) )
			  + id_exFeatures) += *(featureTensor + thread_idx) *
				1.0/sqrt(nodeDegrees[id_exNodes]*
					 nodeDegrees[( (int)*(edgeIndex + numOfDirectedEdges + id_exEdges) )]);
		}
		
	}
	
}

__global__ void GCNLayerNew(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, int numOfNodes, int numOfFeatures, int numOfEdges) {

	int i = threadIdx.x;
	int j = blockIdx. x;
	int k = j;
	//if(i < numOfNodes) {
                //for(int j=0; j<numOfEdges; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(aggregationVar + i*numOfFeatures + k) += *(featureTensor + i*numOfFeatures + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*numOfEdges + j))]);
                                //}
                        }
                //}
                //*(featureTensor + numOfFeatures*i) = *(aggregationVar + i*numOfFeatures);
                //*(featureTensor + numOfFeatures*i + 1) = *(aggregationVar + i*numOfFeatures + 1);
                //*(aggregationVar + i*numOfFeatures) = 0.0;
                //*(aggregationVar + i*numOfFeatures + 1) = 0.0;
        //}


}


} // namespace end
