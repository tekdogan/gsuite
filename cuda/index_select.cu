#include "hip/hip_runtime.h"



void index_select(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize,
                    float *out) {
    int dstTotalSize;
    
    // index along x-axis
    if(dim == 0) {
        dstTotalSize = indSize * srcCols;
    }
    // index along y-axis
    else if(dim == 1){
        dstTotalSize = indSize * srcNums;
    }
    else {
        printf("indexSelect kernel dimension error!\n");
        return;
    }
    
    dim3 largeIndexGrid(dstTotalSize/128);
    dim3 largeIndexBlock(128);
                        
    indexSelectLargeIndex<<<largeIndexGrid,largeIndexBlock>>>
                    (src,srcRows,srcCols,dim,indices,dstTotalSize
                    indSize, out);
    
}

__global__ indexSelectLargeIndex(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize, int dstSize
                    float *out) {
    
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // TODO: calculate the index info below
    const int indexInfo = 1;

    if (thread_idx < dstSize) {

        *(out + threadid_x) = *(src + indexInfo);
    
    }
    
}
