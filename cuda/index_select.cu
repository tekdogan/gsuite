#include "hip/hip_runtime.h"
#include "index_select.h"

float* index_select(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize,
                    float *out) {

    int dstTotalSize;
    
    // calculate output size
    // index along x-axis
    if(dim == 0) {
        dstTotalSize = indSize * srcCols;
    }
    // index along y-axis
    else if(dim == 1){
        dstTotalSize = indSize * srcRows;
    }
    else {
        printf("indexSelect kernel dimension error!\n");
        return src;
    }
    
    // allocate device memory for output
    hipMalloc(&out, dstTotalSize*sizeof(float));

    // dimensions of grids and blocks
    dim3 largeIndexGrid(dstTotalSize/128);
    dim3 largeIndexBlock(128);

    // launch kernel
    indexSelectLargeIndex<<<largeIndexGrid,largeIndexBlock>>>
                    (src,srcRows,srcCols,dim,indices,indSize,
			dstTotalSize,out);

    return out;
}

__global__ void indexSelectLargeIndex(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize, int dstSize,
                    float *out) {
    
	const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_idx < dstSize) {

		// calculate the index info
		const int id_row = (int)*( indices + (int)(thread_idx/indSize) );
		const int id_col = thread_idx % srcCols;

		// update respected cell
		*(out + thread_idx) = *(src + id_row*srcCols + id_col);
	}
    
}
