#include "hip/hip_runtime.h"
#include "index_select.h"

float* index_select(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize) {

	float *d_src, *d_out;
	int *d_indices;

	float *out;
	out = (float*)calloc(indSize*srcCols, sizeof(float));

	hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
	hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**) &d_indices, indSize*sizeof(int));
	hipMemcpy(d_indices, indices, indSize*sizeof(int), hipMemcpyHostToDevice);
	


    int dstTotalSize;
    
    // calculate output size
    // index along x-axis
    if(dim == 0) {
        dstTotalSize = indSize * srcCols;
    }
    // index along y-axis
    else if(dim == 1){
        dstTotalSize = indSize * srcRows;
    }
    else {
        printf("indexSelect kernel dimension error!\n");
        return src;
    }
    
    // allocate device memory for output
    hipMalloc((void**) &d_out, dstTotalSize*sizeof(float));

    // dimensions of grids and blocks
    dim3 largeIndexGrid(dstTotalSize);
    dim3 largeIndexBlock(1);

    // launch kernel
    indexSelectLargeIndex<<<largeIndexGrid,largeIndexBlock>>>
                    (d_src,srcRows,srcCols,dim,d_indices,indSize,
			dstTotalSize,d_out);

    hipMemcpy(out, d_out, dstTotalSize*sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_src);
    hipFree(d_indices);
    hipFree(d_out);
    return out;
}

__global__ void indexSelectLargeIndex(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize, int dstSize,
                    float *out) {
    
	const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_idx < dstSize) {

		// calculate the index info
		const int id_row = (int)*( indices + (int)(thread_idx/srcCols) );
		const int id_col = thread_idx % srcCols;

//		printf("thread id: %d, id_row: %d, id_col, %d\n", thread_idx, id_row, id_col);
		// update respected cell

		int data = *(src + id_row*srcCols + id_col);

		*(out + thread_idx) = data;
	}
    
}
