#include "hip/hip_runtime.h"
#pragma once
#include<hip/hip_runtime.h>

void index_select(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize,
                    float *out) {

    int dstTotalSize;
    
    // calculate output size
    // index along x-axis
    if(dim == 0) {
        dstTotalSize = indSize * srcCols;
    }
    // index along y-axis
    else if(dim == 1){
        dstTotalSize = indSize * srcNums;
    }
    else {
        printf("indexSelect kernel dimension error!\n");
        return;
    }
    
    // allocate device memory for output
    hipMalloc(&out, dstTotalSize*sizeof(float));

    // dimensions of grids and blocks
    dim3 largeIndexGrid(dstTotalSize/128);
    dim3 largeIndexBlock(128);

    // launch kernel
    indexSelectLargeIndex<<<largeIndexGrid,largeIndexBlock>>>
                    (src,srcRows,srcCols,dim,indices,dstTotalSize
                    indSize, out);
    
}

__global__ indexSelectLargeIndex(float *src, int srcRows, int srcCols,
                    int dim, int* indices, int indSize, int dstSize
                    float *out) {
    
	const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_idx < dstSize) {

		// calculate the index info
		const id_row = *( indices + (int)(thread_idx/indSize) );
		const id_col = thread_idx % srcCols;

		// update respected cell
		*(out + threadid_x) = *(src + id_row*srcCols + id_col);
	}
    
}
