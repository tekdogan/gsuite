#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SAG_WL.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"

namespace CU_WL {

__global__ void SAGLayer(float* edgeIndex, float* featureTensor, float w1, float w2, int numOfNodes, int numOfEdges, int numOfFeatures, float* outputFeatureMatrix) {

	printf("thread\n");
	int i = threadIdx.x;
	printf("thread %d\n",i);
	if(i < numOfNodes) {
	    
	        // temporary feature values variable used during
                // the calculation of mean values of incoming edges
                float* tempFeatureValues;
		hipMalloc(&tempFeatureValues, numOfFeatures * sizeof(float));
                        
                // number of incoming edges to i
                int tempIncomingEdges = 0;
	    
	            // scan through edge indices
                for(int j=0; j<numOfEdges; j++) {
                
                        if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
                                
                                // add xj values to sum
                                for(int k=0; k<numOfFeatures; k++) {
                                        *(tempFeatureValues + k) += *(featureTensor + ((int)*(edgeIndex + numOfEdges + j))*numOfFeatures + k);
                                }
                                
                                // increment number of incoming edges to node i
                                tempIncomingEdges++;
                        }
                }
                
                // calculate new values of node features of i
                for(int k=0; k<numOfFeatures; k++) {
			*(outputFeatureMatrix + i*numOfFeatures + k) = (w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (tempFeatureValues[k]/tempIncomingEdges));
			printf("calculated value of node %d feature %d is %f\n",i,k,(w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (tempFeatureValues[k]/tempIncomingEdges)) );
		}
		hipFree(tempFeatureValues);
        }


}

} // namespace end

