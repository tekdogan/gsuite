#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SAG_WL.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"

namespace CU_WL {

__global__ void SAGLayer(float* edgeIndex, float* featureTensor, float w1, float w2, int numOfNodes, int numOfEdges, int numOfFeatures, float* outputFeatureMatrix) {

	int i = threadIdx.x;
	if(i < numOfNodes) {
	    
	            // temporary feature values variable used during
                // the calculation of mean values of incoming edges
                float* tempFeatureValues = (float*)calloc(numOfFeatures, sizeof(float));
                        
                // number of incoming edges to i
                int tempIncomingEdges = 0;
	    
	            // scan through edge indices
                for(int j=0; j<numOfEdges; j++) {
                
                        if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
                                
                                // add xj values to sum
                                for(int k=0; k<numOfFeatures; k++) {
                                        *(tempFeatureValues + k) += *(featureTensor + (*(edgeIndex + numOfEdges + j))*numOfFeatures + k);
                                }
                                
                                // increment number of incoming edges to node i
                                tempIncomingEdges++;
                        }
                }
                
                // calculate new values of node features of i
                
                *(outputFeatureMatrix + i*numOfFeatures + k) = (w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (tempFeatureValues[k]/tempIncomingEdges));
        }


}

} // namespace end
