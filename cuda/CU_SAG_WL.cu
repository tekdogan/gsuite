#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SAG_WL.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"

namespace CU_WL {

__global__ void SAGLayer(float* edgeIndex, float* featureTensor, float w1, float w2, int numOfNodes, int numOfEdges,
			 int numOfFeatures, float* tempFeatureValues, int* tempIncomingEdges, float* outputFeatureMatrix) {

	int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread_idx < numOfNodes*numOfFeatures*numOfDirectedEdges) {
		
		printf("thread_idx is: %d\n", thread_idx);
		
		printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
		
		const int64_t id_exEdges = (thread_idx / numOfNodes * numOfFeatures);
		
		const int64_t id_exNodes = (thread_idx / numOfDirectedEdges * numOfFeatures);
		
		const int64_t id_exFeatures = (thread_idx / numOfNodes * numOfDirectedEdges);
		
		// if an incoming edge to respected node
		if( *(edgeIndex + numOfDirectedEdges + id_exEdges) == id_exNodes ) {
			// apply aggregation of the neighbour node's to temporary
			// feature vector analogus to SAG formula
			*(tempFeatureMatrix + numOfFeatures*( *(edgeIndex + numOfDirectedEdges + id_exEdges) )
				+ id_exFeatures) += *(src + thread_idx);
			
			// increment number of incoming edges to corresponding node
			tempIncomingEdges[id_exNodes]++;
		}
		
		// sync threads before output update
		// __syncthreads();
		
		// update output matrix
		*(outputFeatureMatrix + numOfFeatures*id_exNodes + id_exFeatures) =
			(w1 * *(featureTensor + numOfFeatures*id_exNodes + id_exFeatures)) +
			(w2 * (*(tempFeatureValues + numOfNodes*id_exNodes + id_exFeatures)/tempIncomingEdges[id_exNodes]));
		
	}
	
	// below operations are going to be removed after the
	// kernel update with new computation model
	//if(i < numOfNodes) {
	    
	        // temporary feature values variable used during
                // the calculation of mean values of incoming edges
                //float* tempFeatureValues;
		//hipMalloc(&tempFeatureValues, numOfFeatures * sizeof(float));

		// escape variable to prevent memory dependency
		//bool esc_var = false;
                        
                // number of incoming edges to i
                int tempIncomingEdges = 0;
	        //printf("DEBUG: CU_WL::SAGLayer first part starting...\n");
	        // scan through edge indices
                //for(int j=0; j<numOfEdges; j++) {

                        if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
                                
				//printf("DEBUG: thread[%d] CU_WL::SAGLayer inside incoming edge\n", i);

                                // add xj values to sum
                                //for(int k=0; k<numOfFeatures; k++) {
                                        *(tempFeatureValues + i*numOfNodes + k) += *(featureTensor + ((int)*(edgeIndex + numOfEdges + j))*numOfFeatures + k);
                                //}
                                
                                // increment number of incoming edges to node i
                                tempIncomingEdges++;

				// set escape variable
				//esc_var = true;
                        }
			//else if(esc_var) {
			//	// escape if thread's turn is over
			//	//printf("thread %d escaping at %d\n", i, j);
			//	break;
			//}
			//else {
			//	//printf("DEBUG: else\n");
			//}
                //}
                //printf("DEBUG: CU_WL::SAGLayer THREAD:%d first part successful!\n", i);

		//printf("DEBUG: numOfFeatures is %d\n", numOfFeatures);
                // calculate new values of node features of i
                //for(int k=0; k<numOfFeatures; k++) {
			*(outputFeatureMatrix + i*numOfFeatures + k) = (w1 * *(featureTensor + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEdges));
			//printf("DEBUG: CU_WL::SAGLayer inside the aggregation part.\n");
			//printf("DEBUG: CU_WL::SAGLayer THREAD:%d calculated value of node %d feature %d is %f\n",i,i,k,(w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEdges)) );
		//}
		//hipFree(tempFeatureValues);
		
        //}


}


__global__ void SAGLayer2(float* edgeIndex, float* featureTensor, float w1, float w2, int numOfNodes, int numOfEdges, int numOfFeatures, float* tempFeatureValues, float* outputFeatureMatrix) {

        int i = threadIdx.x + blockIdx.x * blockDim.x;
        //printf("thread %d\n",i);
        if(i < numOfNodes) {

                // temporary feature values variable used during
                // the calculation of mean values of incoming edges
                //float* tempFeatureValues;
                //hipMalloc(&tempFeatureValues, numOfFeatures * sizeof(float));

                // number of incoming edges to i
                int tempIncomingEdges = 0;
                //printf("DEBUG: CU_WL::SAGLayer first part starting...\n");
                // scan through edge indices
                for(int j=0; j<numOfEdges; j++) {

                        if((*(edgeIndex + j*2)) == (float)i) { // if there is an edge incoming to node i

                                // add xj values to sum
                                for(int k=0; k<numOfFeatures; k++) {
                                        *(tempFeatureValues + i*numOfNodes + k) += *(featureTensor + ((int)*(edgeIndex + 2*j))*numOfFeatures + k);
                                }

                                // increment number of incoming edges to node i
                                tempIncomingEdges++;
                        }
                }
                //printf("DEBUG: CU_WL::SAGLayer THREAD:%d first part successful!\n", i);

                //printf("DEBUG: numOfFeatures is %d\n", numOfFeatures);
                // calculate new values of node features of i
                for(int k=0; k<numOfFeatures; k++) {
                        *(outputFeatureMatrix + i*numOfFeatures + k) = (w1 * *(featureTensor + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEdges));
                        //printf("DEBUG: CU_WL::SAGLayer inside the aggregation part.\n");
                        //printf("DEBUG: CU_WL::SAGLayer THREAD:%d calculated value of node %d feature %d is %f\n",i,i,k,(w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEd$
                }
                //hipFree(tempFeatureValues);

        }


}

} // namespace end

