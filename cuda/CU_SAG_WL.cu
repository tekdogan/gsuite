#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "CU_SAG_WL.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"

namespace CU_WL {

__global__ void SAGLayer(float* edgeIndex, float* featureTensor, float w1, float w2, int numOfNodes, int numOfEdges, int numOfFeatures, float* tempFeatureValues, float* outputFeatureMatrix) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("thread %d\n",i);
	if(i < numOfNodes) {
	    
	        // temporary feature values variable used during
                // the calculation of mean values of incoming edges
                //float* tempFeatureValues;
		//hipMalloc(&tempFeatureValues, numOfFeatures * sizeof(float));
                        
                // number of incoming edges to i
                int tempIncomingEdges = 0;
	        //printf("DEBUG: CU_WL::SAGLayer first part starting...\n");
	        // scan through edge indices
                for(int j=0; j<numOfEdges; j++) {
                
                        if((*(edgeIndex + j)) == (float)i) { // if there is an edge incoming to node i
                                
                                // add xj values to sum
                                for(int k=0; k<numOfFeatures; k++) {
                                        *(tempFeatureValues + i*numOfNodes + k) += *(featureTensor + ((int)*(edgeIndex + numOfEdges + j))*numOfFeatures + k);
                                }
                                
                                // increment number of incoming edges to node i
                                tempIncomingEdges++;
                        }
                }
                //printf("DEBUG: CU_WL::SAGLayer THREAD:%d first part successful!\n", i);

		//printf("DEBUG: numOfFeatures is %d\n", numOfFeatures);
                // calculate new values of node features of i
                for(int k=0; k<numOfFeatures; k++) {
			*(outputFeatureMatrix + i*numOfFeatures + k) = (w1 * *(featureTensor + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEdges));
			//printf("DEBUG: CU_WL::SAGLayer inside the aggregation part.\n");
			//printf("DEBUG: CU_WL::SAGLayer THREAD:%d calculated value of node %d feature %d is %f\n",i,i,k,(w1 * *(outputFeatureMatrix + i*numOfFeatures + k)) + (w2 * (*(tempFeatureValues + i*numOfNodes + k)/tempIncomingEdges)) );
		}
		//hipFree(tempFeatureValues);
		
        }


}

} // namespace end

