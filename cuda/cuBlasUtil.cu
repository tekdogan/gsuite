#include "hip/hip_runtime.h"
#include<hipblas.h>
#include"cuBlasUtil.h"
#include<stdio.h>

void gpu_blas_mmul(const float *A, const float *B, float *C, int m, int n, int k, bool transA, bool transB, float Alpha, float Beta) {

        int lda=m,ldb=k,ldc=m;
        const float alf = Alpha;
        const float bet = Beta;
        const float *alpha = &alf;
        const float *beta = &bet;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

	hipblasOperation_t tA, tB;
	if(transA) {
		tA = HIPBLAS_OP_T;
	}
	else {
		tA = HIPBLAS_OP_N;
	}
	if(transB) {
                tB = HIPBLAS_OP_T;
        }
        else {
                tB = HIPBLAS_OP_N;
        }

	// plain sgemm
	//hipblasStatus_t status = hipblasSgemm(handle, tA, tB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	
	// sgemm with precisions indicated
	hipblasStatus_t status = cublasSgemmEx(handle, tA, tB, m, n, k, alpha, A, HIP_R_32F, lda, B, HIP_R_32F, ldb, beta, C, HIP_R_32F, ldc);
	
	// batched sgemm
	//hipblasStatus_t status = hipblasSgemmBatched(handle, tA, tB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, 32);

	hipDeviceSynchronize();

	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS operation failed\n");
	}
	else {
		printf("CUBLAS operation is successful!\n");
	}

        hipblasDestroy(handle);
}


__global__ void initIdentityGPU(float **devMatrix, int numR, int numC) {
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if(y < numR && x < numC) {
          if(x == y)
              devMatrix[y][x] = 1;
          else
              devMatrix[y][x] = 0;
    }
}


void initIdentityMatrix(float* matrix, int R, int C) {

	for(int i=0; i<R; i++) {
		for(int j=0; j<C; j++) {
			if(i == j) {
				*( matrix + i*C + j ) = 1.0;
			}
			else {
				*( matrix + i*C + j ) = 0.0;
			}
		}
	}

}
