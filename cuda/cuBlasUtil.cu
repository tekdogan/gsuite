#include "hip/hip_runtime.h"
#include<hipblas.h>
#include"cuBlasUtil.h"
#include<stdio.h>

void gpu_blas_mmul(const float *A, const float *B, float *C, int m, int n, int k, bool transA, bool transB, float alpha, float beta) {

        int lda=m,ldb=k,ldc=m;
        const float alf = alpha;
        const float bet = beta;
        const float *alpha = &alf;
        const float *beta = &bet;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

	hipblasOperation_t tA, tB;
	if(transA) {
		tA = HIPBLAS_OP_T;
	}
	else {
		tA = HIPBLAS_OP_N;
	}
	if(transB) {
                tB = HIPBLAS_OP_T;
        }
        else {
                tB = HIPBLAS_OP_N;
        }

	hipblasStatus_t status = hipblasSgemm(handle, tA, tB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS operation failed\n");
	}
	else {
		printf("CUBLAS operation is successful!\n");
	}

        hipblasDestroy(handle);
}


__global__ void initIdentityGPU(int **devMatrix, int numR, int numC) {
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if(y < numR && x < numC) {
          if(x == y)
              devMatrix[y][x] = 1;
          else
              devMatrix[y][x] = 0;
    }
}
