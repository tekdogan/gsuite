#include<hipblas.h>
#include"cuBlasUtil.h"
#include<stdio.h>

void gpu_blas_mmul(const float *A, const float *B, float *C, int m, int n, int k, bool transA, bool transB) {

        int lda=m,ldb=k,ldc=m;
        const float alf = 1;
        const float bet = 0;
        const float *alpha = &alf;
        const float *beta = &bet;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

	hipblasOperation_t tA, tB;
	if(transA) {
		tA = HIPBLAS_OP_T;
	}
	else {
		tA = HIPBLAS_OP_N;
	}
	if(transB) {
                tB = HIPBLAS_OP_T;
        }
        else {
                tB = HIPBLAS_OP_N;
        }

	hipblasStatus_t status = hipblasSgemm(handle, tA, tB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS operation failed\n");
	}
	else {
		printf("CUBLAS operation is successful!\n");
	}

        hipblasDestroy(handle);
}

