#include "hip/hip_runtime.h"
/*
* 
* Nvidia's Simple Quick Sort Sample was used as a basis and modified to meet our
* CPU algorithm. Check CUDA TOOLKIT 6.0 SAMPLES.
*
* As requested by NVIDIA:
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
* =============================================================================
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
* =============================================================================
* Merge sort written by us, based on a top down implementation of merge sort
*
* Eugenio Pacceli Reis da Fonseca
* Ícaro Pinto Coelho Harry
* DCC/UFMG 2014
* Compiler:: NVCC v6.0.1
* Debian testing/sid 64 bits
*/

#include "hip/hip_runtime_api.h"
#include "helper_string.h"

//extern "C" {
    #include "cuda_sort.h"
//}

// NVIDIA's
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
__device__ void selection_sort( int *data, int left, int right, int edgeSize)
{
  for( int i = left ; i <= right ; ++i ){
    int min_val = data[i];
    int min_idx = i;

    int min_val_ind = *(data+i+edgeSize);
    //printf("DEBUG: i: %d, min_val_ind: %d\n", i, min_val_ind);

    // Find the smallest value in the range [left, right].
    for( int j = i+1 ; j <= right ; ++j ){
      int val_j = data[j];
      int val_j_ind = data[j+edgeSize];
      if( val_j < min_val ){
        min_idx = j;
        min_val = val_j;
	min_val_ind = val_j_ind;
      }
    }

    // Swap the values.
    if( i != min_idx ){
      data[min_idx] = data[i];
      data[min_idx + edgeSize] = data[i + edgeSize];
      data[i] = min_val;
      data[i + edgeSize] = min_val_ind;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(int *data, int left, int right, int depth, int edgeSize){
    //If we're too deep or there are few elements left, we use an insertion sort...
    if( depth >= MAX_DEPTH || right-left <= INSERTION_SORT ){
        selection_sort( data, left, right, edgeSize);
        return;
    }


    hipStream_t s,s1;
    int *lptr = data+left;
    int *rptr = data+right;
    int  pivot = data[(left+right)/2];

    int lval;
    int rval;

    int lvalind, rvalind;

    int nright, nleft;

    // Do the partitioning.
    while (lptr <= rptr){
        // Find the next left- and right-hand values to swap
        lval = *lptr;
	lvalind = *(lptr + edgeSize);
        rval = *rptr;
	rvalind = *(rptr + edgeSize);

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot && lptr < data+right){
            lptr++;
            lval = *lptr;
	    lvalind = *(lptr + edgeSize);
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot && rptr > data+left){
            rptr--;
            rval = *rptr;
	    rvalind = *(rptr + edgeSize);
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr){
            *lptr = rval;
            *(lptr + edgeSize) = rvalind;
            *rptr = lval;
            *(rptr + edgeSize) = lvalind;
            lptr++;
            rptr--;
        }
    }

    // Now the recursive part
    nright = rptr - data;
    nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data)){
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1, edgeSize);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right){
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1, edgeSize);
        hipStreamDestroy(s1);
    }
}


// gcc compiled code will call this function to access CUDA Quick Sort.
// This calls the kernel, which is recursive. Waits for it, then copies it's
// output back to CPU readable memory.
//extern "C"
void gpu_qsort(int *data, int n, int edgeSize){
    int* gpuData;
    int left = 0;
    int right = n-1;

    // Prepare CDP for the max depth 'MAX_DEPTH'.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    // Allocate GPU memory.
    hipMalloc((void**)&gpuData,2*n*sizeof(int));
    hipMemcpy(gpuData,data, 2*n*sizeof(int), hipMemcpyHostToDevice);

    // Launch on device
    cdp_simple_quicksort<<< 1, 1 >>>(gpuData, left, right, 0, edgeSize);
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(data,gpuData, 2*n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpuData);
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}

// Our MergeSort implementation. Really simple, based on the normal top down
// implementation. Not inplace.
__global__ void simple_mergesort(int* data,int *dataAux,int begin,int end, int depth){
    int middle = (end+begin)/2;
    int i0 = begin;
    int i1 = middle;
    int index;
    int n = end-begin;

    // Used to implement recursions using CUDA parallelism.
    hipStream_t s,s1;

    if(n < 2){
        return;
    }

    // Launches a new block to sort the left part.
    hipStreamCreateWithFlags(&s,hipStreamNonBlocking);
    simple_mergesort<<< 1, 1, 0, s >>>(data,dataAux, begin, middle, depth+1);
    hipStreamDestroy(s);

    // Launches a new block to sort the right part.
    hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);
    simple_mergesort<<< 1, 1, 0, s1 >>>(data,dataAux, middle, end, depth+1);
    hipStreamDestroy(s1);

    // Waits for children's work.
    hipDeviceSynchronize();

    // Merges children's generated partition.
    // Does the merging using the auxiliary memory.
    for (index = begin; index < end; index++) {
        if (i0 < middle && (i1 >= end || data[i0] <= data[i1])){
            dataAux[index] = data[i0];
            i0++;
        }else{
            dataAux[index] = data[i1];
            i1++;
        }
    }

    // Copies from the auxiliary memory to the main memory.
    // Note that each thread operates a different partition,
    // and the auxiliary memory has exact the same size of the main
    // memory, so the threads never write or read on the same
    // memory position concurrently, since one must wait it's children
    // to merge their partitions.
    for(index = begin; index < end; index++){
        data[index] = dataAux[index];
    }
}

// gcc compiled code will call this function to access CUDA Merge Sort.
// This calls the kernel, which is recursive. Waits for it, then copies it's
// output back to CPU readable memory.
//extern "C"
void gpumerge_sort(int* a,int n){
    int* gpuData;
    int* gpuAuxData;
    int left = 0;
    int right = n;

    // Prepare CDP for the max depth 'MAX_DEPTH'.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    // Allocate GPU memory.
    hipMalloc((void**)&gpuData,n*sizeof(int));
    hipMalloc((void**)&gpuAuxData,n*sizeof(int));
    hipMemcpy(gpuData,a, n*sizeof(int), hipMemcpyHostToDevice);

    // Launch on device
    simple_mergesort<<< 1, 1 >>>(gpuData,gpuAuxData, left, right, 0);
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(a,gpuData, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpuAuxData);
    hipFree(gpuData);
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}

