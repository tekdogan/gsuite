#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime.h>
#include "CU_SpMM_GCN.h"
#include "cuBlasUtil.h"
#include "Data_Util.h"


namespace CU_SpMM {


void GCNLayer(float* adjMatrix, float* featureTensor, int n_nodes, int n_edges, int n_features, float* output) {

//    int i = threadIdx.x;
//    if(i < n_nodes) {
        
        
        // ----- calculation of A^ ----- //
	for(int i=0; i<n_nodes; i++) {
		// add self loops
		*(adjMatrix + (n_nodes+1)*i) += 1.0;
	}

	printf("A matrix:\n");
	printDenseMatrix(adjMatrix, n_nodes, n_nodes);

	// define device matrices
        float *d_A, *d_D, *d_DA, *d_DAD, *d_DADX, *d_X;

	// allocate device A and D matrices
        hipMalloc(&d_A,n_nodes * n_nodes * sizeof(float));
        hipMalloc(&d_D,n_nodes * n_nodes * sizeof(float));
                
	// ----- calculation of D^-1/2 ----- //
	float* D = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	for(int i=0; i<n_nodes; i++) {
		for(int j=0; j<=i; j++) {
		*(D + (n_nodes+1)*i) += *(adjMatrix + i*n_nodes + j);
	}
	// square root
	*(D + (n_nodes+1)*i) = sqrt((int)*(D + (n_nodes+1)*i));
	}

	printf("D matrix:\n");
	printDenseMatrix(D, n_nodes, n_nodes);


	// migrate A and D matrices to device
        hipMemcpy(d_A,adjMatrix,n_nodes * n_nodes * sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_D,D,n_nodes * n_nodes * sizeof(float),hipMemcpyHostToDevice);
        

	// ----- calculation of D^-1/2 * A^ ----- //
	hipMalloc(&d_DA,n_nodes * n_nodes * sizeof(float));
	gpu_blas_mmul(d_A, d_D, d_DA, n_nodes, n_nodes, n_nodes, false, false);
	float* DA = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	//gpu_blas_mmul(D, adjMatrix, DA, n_nodes, n_nodes, n_nodes);
	hipFree(d_A);

	hipMemcpy(DA,d_DA,n_nodes * n_nodes * sizeof(float),hipMemcpyDeviceToHost);
	printf("DA matrix:\n");
	printDenseMatrix(DA, n_nodes, n_nodes);
        
	// ----- calculation of D^-1/2 * A^ * D^-1/2 ----- //
	hipMalloc(&d_DAD,n_nodes * n_nodes * sizeof(float));
	gpu_blas_mmul(d_D, d_DA, d_DAD, n_nodes, n_nodes, n_nodes, false, false);
        hipFree(d_DA);
        hipFree(d_D);

	float* DAD = (float*)calloc(n_nodes*n_nodes, sizeof(float));
	hipMemcpy(DAD,d_DAD,n_nodes * n_nodes * sizeof(float),hipMemcpyDeviceToHost);
	printf("DAD matrix:\n");
	printDenseMatrix(DAD, n_nodes, n_nodes);
        
	// migrate node feature values from host to device
	hipMemcpy(d_X, featureTensor, n_nodes * n_features * sizeof(float), hipMemcpyHostToDevice);

	printf("X matrix:\n");
	printDenseMatrix(featureTensor, n_nodes, n_features);

	// ----- calculation of D^-1/2 * A^ * D^-1/2 * X ----- //
	hipMalloc(&d_DADX,n_nodes * n_features * sizeof(float));
	gpu_blas_mmul(d_X, d_DAD, d_DADX, n_features, n_nodes, n_features, false, false);
	hipFree(d_DAD);
	hipFree(d_X);

	// copy the result to output
	float* DADX = (float*)calloc(n_nodes * n_features, sizeof(float));
	hipMemcpy(DADX, d_DADX, n_nodes * n_features * sizeof(float), hipMemcpyDeviceToHost);
	memcpy(output, DADX, sizeof(float)*(n_nodes*n_features));

	printf("DADX matrix:\n");
	printDenseMatrix(DADX, n_nodes, n_features);
//    }

}

} //namespace end
