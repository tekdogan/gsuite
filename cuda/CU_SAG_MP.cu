#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include"CU_SAG_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

#define DIRECTED_EDGES 4

#define NUM_NODES 3

#define FEATURE_LEN 3


namespace CU_MP {

__global__ void SAGLayer(float* edgeIndex, float* featureTensor, float *aggregationVar, float *nodeDegrees, float w1, float w2) {

	int i = threadIdx.x;
	if(i < NUM_NODES) {
                for(int j=0; j<DIRECTED_EDGES; j++) {
                        if((*(edgeIndex + j)) == (float)i) {// if there is an edge incoming to node i
                                // aggregate edgeIndex[1][j] features on node i
                                //std::cout << "from node " << edgeIndex[1][j] << " to node " << i << std::endl;
                                for(int k=0; k<FEATURE_LEN; k++) {
                                        aggregationVar[k] += *(featureTensor + i*FEATURE_LEN + k) * 1.0/sqrt(nodeDegrees[i]*nodeDegrees[(int)(*(edgeIndex + 1*DIRECTED_EDGES + j))]);
                                }
                        }
                }
                *(featureTensor + FEATURE_LEN*i) = aggregationVar[0];
                *(featureTensor + FEATURE_LEN*i + 1) = aggregationVar[1];
                aggregationVar[0] = 0.0;
                aggregationVar[1] = 0.0;
        }


}


} // namespace end
