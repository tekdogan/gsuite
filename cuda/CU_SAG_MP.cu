#include<iostream>
#include<cmath>
#include"CU_SAG_MP.h"
#include<omp.h>
#include<hip/hip_runtime.h>

#include "index_select.h"
#include "scatter_cuda.h"
#include "linear.h"

namespace CU_MP {

float* SAGELayer(int* h_edgeIndex, float* h_featureVector, int numOfNodes, int numOfFeatures,
                        int numOfEdges, int outputSize) {


        int* h_edgeIndexSrc;
        int* h_edgeIndexDst;
        h_edgeIndexSrc = (int*)calloc(numOfEdges, sizeof(int));
        h_edgeIndexDst = (int*)calloc(numOfEdges, sizeof(int));

        memcpy(h_edgeIndexSrc, h_edgeIndex, numOfEdges);
        memcpy(h_edgeIndexDst, h_edgeIndex + numOfEdges, numOfEdges);


        float* indexSelectOutput = index_select(h_featureVector, numOfNodes, numOfFeatures, 0, h_edgeIndexSrc, numOfEdges);

        float* aggrOutput = scatter_cuda(indexSelectOutput, h_edgeIndexDst, 1, "sum", numOfEdges, numOfEdges, numOfFeatures, numOfNodes, numOfFeatures);

        float* aggrTransformed = linear(aggrOutput, numOfNodes, numOfFeatures, numOfNodes, outputSize);

	float* inputTransformed = linear(h_featureVector, numOfNodes, numOfFeatures, numOfNodes, outputSize);

	for(int i = 0; i < numOfNodes*outputSize; i++) {
		aggrTransformed[i] += inputTransformed[i];
	}

        free(h_edgeIndexSrc);
        free(h_edgeIndexDst);
        free(indexSelectOutput);
	free(inputTransformed);

        return aggrTransformed;	
}


} // namespace end
