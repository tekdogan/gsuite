#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

#include "reducer.cuh"
//#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const float *src_data, float *out_data,
               int numOfNodes, int numOfFeatures, int numOfEdges) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numOfNodes*numOfFeatures*numOfEdges) {
    
    // get indices of the thread
    
    int64_t idx = (thread_idx / numOfEdges);
    
    int64_t index_info = thread_idx % (numOfFeatures*numOfEdges);
    
    int64_t id_r = (idx / numOfNodes);
    
    int64_t id_c = (id_r / numOfFeatures);

    if(( *(edgeIndex + edgeIndexSize + index_info) == id_c)) { // an incoming edge to node id_r
        Reducer<scalar_t, REDUCE>::atomic_write(out_data + id_r * numOfFeatures + id_c,
                                            src_data + id_r * numOfFeatures + id_c);
    }
  }
}

/*template <typename scalar_t>
__global__ void
scatter_arg_kernel(float *src_data, float *out_data, int64_t *arg_out_data,
                    int E, int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    if (src_data[thread_idx] == out_data[b * N * K + idx * K + k]) {
      arg_out_data[b * N * K + idx * K + k] = e;
    }
  }
}*/

std::tuple<float*, float*>
scatter_cuda(float *src, float *index, int64_t dim,
             std::string reduce, int numOfNodes, int numOfFeatures,
             int numOfEdges) {
  
  CHECK_CUDA(src);
  CHECK_CUDA(index);
  
  hipSetDevice(0);
  
  float* out = calloc(edgeIndexSize * 2, sizeof(float));
  
  float* arg_out = calloc(edgeIndexSize * 2, sizeof(float));
  
  int64_t *arg_out_data = nullptr;
  
  
  // pick dim = 1
  auto B = numOfNodes; // mul of each dimension less than dim
  auto E = numOfFeatures; // size of dimension in dim
  auto K = 1; // mul of each dimension greater than dim
  auto N = numOfFeatures; // output size of dim


    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      //if (!optional_out.has_value())
      //  out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<scalar_t, REDUCE>
          <<<BLOCKS(numOfNodes*numOfFeatures*numOfEdges), THREADS, 0, stream>>>(
              src_data, out_data, numOfNodes, numOfFeatures, numOfEdges);

      //if (!optional_out.has_value() && (REDUCE == MIN || REDUCE == MAX))
      //  out.masked_fill_(out == Reducer<scalar_t, REDUCE>::init(), (scalar_t)0);

      if (REDUCE == MIN || REDUCE == MAX)
        scatter_arg_kernel<scalar_t>
            <<<BLOCKS(numOfNodes), THREADS, 0, stream>>>(
                src_data, out_data, numOfNodes, numOfFeatures,
                numOfEdges);
    });
  });

  return std::make_tuple(out, arg_out);
}
