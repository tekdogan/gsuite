#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const float *src_data, const int *indices, float *out_data,
               int numOfRows, int numOfColumns, int indSize, int dim) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // TO DO: size of src_data
  if (thread_idx < indSize) {

    Reducer<scalar_t, REDUCE>::atomic_write(out_data + (int)*(indices + thread_idx),
                                            *(src_data + thread_idx));
  }
}


float* scatter_cuda(float *h_src, int *h_index, int64_t dim,
             std::string reduce, int indSize, int srcRows,
             int srcCols, int outRows, int outCols) {
    
  hipSetDevice(0);
  
  float *d_src;
  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

  int *d_index;
  hipMalloc((void**) &d_index, indSize*sizeof(int));
  hipMemcpy(d_index, h_index, indSize*sizeof(int), hipMemcpyHostToDevice);  

  float *d_out;
  hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));
  

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      //if (!optional_out.has_value())
      //  out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<float, REDUCE>
          <<<BLOCKS(srcRows*srcCols), THREADS>>>(
              d_src, d_index, d_out, srcRows, srcCols,
		indSize, dim);

    });

  float *h_out = (float*)calloc(outRows*outCols, sizeof(float));
  hipMemcpy(h_out, d_out, outRows*outCols*sizeof(float), hipMemcpyDeviceToHost);

  return h_out;
}
