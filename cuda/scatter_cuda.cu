#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const float *src_data, const int *indices, float *out_data,
               int numOfRows, int numOfColumns, int indSize, int dim) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int input_size = numOfRows*numOfColumns;
  // TO DO: size of src_data
  if (thread_idx < input_size) {

    int src_r = thread_idx / numOfColumns;
    int out_r = indices[src_r];

    int src_c = thread_idx % numOfColumns;
    int out_c = src_c;

//    printf("out data index: %d\n, out_r: %d, src_r: %d, indSize: %d\n", out_r*numOfColumns + out_c, out_r, src_r, indSize);
    const float* address = src_data + src_r*numOfColumns + src_c;
    
    float data =  *(address);


    Reducer<scalar_t, REDUCE>::atomic_write(out_data + out_r*numOfColumns + out_c,
                                            data);

  }
}


float* scatter_cuda(float *h_src, int *h_index, int64_t dim,
             std::string reduce, int indSize, int srcRows,
             int srcCols, int outRows, int outCols) {
    
  hipSetDevice(0);
  
  float *d_src;
  hipError_t e = hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  const char* err = hipGetErrorString(e);

  e = hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  err = hipGetErrorString(e);


  int *d_index;
  e = hipMalloc((void**) &d_index, indSize*sizeof(int));
  err = hipGetErrorString(e);
  

  e = hipMemcpy(d_index, h_index, indSize*sizeof(int), hipMemcpyHostToDevice);  
  err = hipGetErrorString(e);


  float *d_out;
  e = hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));
  err = hipGetErrorString(e);


 printf("out max size: %d\n", outRows*outCols);

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      //if (!optional_out.has_value())
      //  out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<float, REDUCE>
          <<<BLOCKS(srcRows*srcCols), THREADS>>>(
              d_src, d_index, d_out, srcRows, srcCols,
		indSize, dim);

    });

  printf("debug scatter kernel launched\n");

  hipDeviceSynchronize();
  float *h_out = (float*)calloc(outRows*outCols, sizeof(float));
  e = hipMemcpy(h_out, d_out, outRows*outCols*sizeof(float), hipMemcpyDeviceToHost);
  err = hipGetErrorString(e);

  e = hipFree(d_out);
  err = hipGetErrorString(e);

  e = hipFree(d_src);
  err = hipGetErrorString(e);
  
  e = hipFree(d_index);
  err = hipGetErrorString(e);

  return h_out;
}
