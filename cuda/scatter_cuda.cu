#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const float *src_data, const *indices, float *out_data,
               int numOfRows, int numOfColumns, int indSize, int dim) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  \\ TO DO: size of src_data
  if (thread_idx < numOfNodes*numOfFeatures) {
    
    Reducer<scalar_t, REDUCE>::atomic_write(out_data + (int)*(indices + thread_idx),
                                            src_data[thread_idx]);
  }
}


float* scatter_cuda(float *h_src, int *h_index, int64_t dim,
             std::string reduce, int indSize, int srcRows,
             int srcCols) {
    
  hipSetDevice(0);
  
  int64_t *arg_out_data = nullptr;

  float *d_src;
  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  hipMemcpy(d_src, h_src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);

  int *d_index;
  hipMalloc((void**) &d_index, indSize*sizeof(int));
  hipMemcpy(d_index, h_index, indSize*sizeof(int), hipMemcpyHostToDevice);  

  float *d_out;
  hipMalloc((void**) &d_out, srcRows*srcCols*sizeof(float));
  

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      //if (!optional_out.has_value())
      //  out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<float, REDUCE>
          <<<BLOCKS(srcRows*srcCols), THREADS>>>(
              d_src, d_index, d_out, srcRows, srcCols,
		indSize, dim);

    });

  return out;
}
