#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

#include "reducer.cuh"
//#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const float *src_data, float *out_data,
               int numOfRows, int numOfColumns, int dim) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  \\ TO DO: size of src_data
  if (thread_idx < numOfNodes*numOfFeatures) {
    
    // get indices of the thread
    
    int64_t id_c = (thread_idx / numOfRows);
    
    int64_t id_r = (thread_idx / numOfColumns);
    
    Reducer<scalar_t, REDUCE>::atomic_write(out_data + id_r * numOfColumns + id_c,
                                            src_data[thread_idx]);
  }
}

/*template <typename scalar_t>
__global__ void
scatter_arg_kernel(float *src_data, float *out_data, int64_t *arg_out_data,
                    int E, int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    if (src_data[thread_idx] == out_data[b * N * K + idx * K + k]) {
      arg_out_data[b * N * K + idx * K + k] = e;
    }
  }
}*/

float* scatter_cuda(float *src, float *index, int64_t dim,
             std::string reduce, int numOfNodes, int numOfFeatures,
             int numOfEdges) {
  
//  CHECK_CUDA(src);
//  CHECK_CUDA(index);
  
  hipSetDevice(0);
  
  float* out = (float*) calloc(numOfEdges * 2, sizeof(float));
  
  float* arg_out = (float*) calloc(numOfEdges * 2, sizeof(float));
  
  int64_t *arg_out_data = nullptr;
  
  
  // pick dim = 1
  auto B = numOfNodes; // mul of each dimension less than dim
  auto E = numOfFeatures; // size of dimension in dim
  auto K = 1; // mul of each dimension greater than dim
  auto N = numOfFeatures; // output size of dim


    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      //if (!optional_out.has_value())
      //  out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<float, REDUCE>
          <<<BLOCKS(numOfNodes*numOfFeatures), THREADS>>>(
              src, out, numOfNodes, numOfFeatures, numOfEdges);

      //if (!optional_out.has_value() && (REDUCE == MIN || REDUCE == MAX))
      //  out.masked_fill_(out == Reducer<scalar_t, REDUCE>::init(), (scalar_t)0);
/*
      if (REDUCE == MIN || REDUCE == MAX)
        scatter_arg_kernel<scalar_t>
            <<<BLOCKS(numOfNodes), THREADS, 0, stream>>>(
                src_data, out_data, numOfNodes, numOfFeatures,
                numOfEdges);
*/
    });


  return out;
}
