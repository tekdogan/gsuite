#include"DataLoader.h"

#define SIZE 1024

int main() {

	float *edgeIndex, *featureVector;

	float aggregationVar[2] = {0,0};
	float nodeDegrees[3] = {1,2,1};

	const char* edgeIndexFileName = "cora.cites";
	int edgeIndexSize = getEdgeIndexSizeFromFile(edgeIndexFileName);
	std::cout << "edgeIndexSize: " << edgeIndexSize << std::endl;

	const char* featureFileName = "cora.content";
	int featureSize = getFeatureSizeFromFile(featureFileName);
	std::cout << "featureSize: " << featureSize << std::endl;

	int numOfNodes = getNumOfNodesFromFile(featureFileName);
	std::cout << "numOfNodes: " << numOfNodes << std::endl;

	try {
		hipMallocManaged( (void**) &edgeIndex, (unsigned long) featureSize*edgeIndexSize * sizeof(float));
		loadEdgeIndexFromFile(edgeIndexFileName, edgeIndex, edgeIndexSize);
	} catch(...) {
		std::cout << "Could not allocate memory space for edgeIndex!\n";
	}

	try {
		hipMallocManaged( (void**) &featureVector, numOfNodes*featureSize * sizeof(float));
		loadFeatureVectorFromFile(featureFileName, featureVector, featureSize);
	} catch(...) {
		std::cout << "Could not allocate memory space for featureVector!\n";
	}

	auto start = std::chrono::steady_clock::now();

	CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees);

	CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees);

	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "2-layer GCN execution took " << dur_ms.count() << " ms\n";

	for(int i=0; i<3; i++) {
		std::cout << "Node" << i << " feature 0: " << *(featureVector + featureSize*i) << std::endl;
		std::cout << "Node" << i << " feature 1: " << *(featureVector + featureSize*i + 1) << std::endl;
	}


	float *featureVectorOutput;
	hipMallocManaged( (void**) &featureVectorOutput, numOfNodes*featureSize * sizeof(float));
        start = std::chrono::steady_clock::now();
        //GINLayer<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, 1.0, featureVectorOutput);
        end = std::chrono::steady_clock::now();
        dur_ms = end-start;
        std::cout << "1-layer GIN execution took " << dur_ms.count() << " ms\n";

	for(int i=0; i<3; i++) {
                std::cout << "Node" << i << " feature 0: " << *(featureVectorOutput + featureSize*i) << std::endl;
                std::cout << "Node" << i << " feature 1: " << *(featureVectorOutput + featureSize*i + 1) << std::endl;
        }

	hipFree(edgeIndex);
	hipFree(featureVector);
	hipFree(featureVectorOutput);

	return 0;
}

int getEdgeIndexSizeFromFile(const char* fileName) {

	std::ifstream dsFile(fileName);
	std::string line;

	int numOfEdges = 0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			numOfEdges++;
		}
	}
	else {
		std::cout << "Could not open the edgeIndex dataset file!\n";
		return -1;
	}

	return numOfEdges;

}

void loadEdgeIndexFromFile(const char* fileName, float* edgeIndex, const int numOfEdges) {

	std::ifstream dsFile(fileName);
	std::string line;

	int i=0, j=0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			std::istringstream ss(line);
			std::string word;
			while(std::getline(ss, word, ' ')) {
				*(edgeIndex + (numOfEdges*i) + j) = std::stof(word);
				i = 1;
			}
		j++;
		i = 0;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
	}

}

int getFeatureSizeFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
	std::string line, word;

	int numOfFeatures = -1;

	if(dsFile.is_open()) {
		getline(dsFile, line);
		std::istringstream ss(line);
		while(std::getline(ss, word, ' ')) {
			numOfFeatures++;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
		return -1;
	}

	return numOfFeatures;
}

int getNumOfNodesFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
        std::string line;

        int numOfNodes = 0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
			numOfNodes++;
		}
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
                return -1;
        }

        return numOfNodes;

}

void loadFeatureVectorFromFile(const char* fileName, float* featureVector, int featureSize) {
	std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
                        std::istringstream ss(line);
                        std::string word;
			std::getline(ss, word, ' '); // escape node index
                        while(std::getline(ss, word, ' ')) {
                                *(featureVector + i*featureSize + j) = std::stof(word);
                                j += 1;
                        }
                i++;
                j = 0;
                }
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
        }
}
