#include "linear.h"

float* linear(float *src, int srcRows, int srcCols, int outRows, int outCols) {
  
  float *w, *d_src, *d_out;
  
  float *out;
  out = (float*)calloc(outRows*outCols, sizeof(float));

  // allocate device memory for output
  hipError_t e = hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));

  const char* err = hipGetErrorString(e);

  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  e = hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  err = hipGetErrorString(e);  


  // allocate device memory for weight
  hipMalloc((void**) &w, srcCols*outCols*sizeof(float));

  float *h_w = (float*)calloc(srcCols*outCols, sizeof(float));
  memset(h_w, 1, srcCols*outCols*sizeof(float));
  e = hipMemcpy(w, h_w, srcCols*outCols*sizeof(float), hipMemcpyHostToDevice);
  err = hipGetErrorString(e);

  // init weight matrix
  //initIdentityGPU<<<srcCols*outCols,1>>>(&w, srcCols, outCols);
  
  gpu_blas_mmul(w, d_src, d_out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);

  hipDeviceSynchronize();

  e = hipMemcpy(out,d_out,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  err = hipGetErrorString(e);

  e = hipFree(d_out);
  hipFree(w);
  hipFree(d_src);
  
  return out;
}
