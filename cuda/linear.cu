#include "linear.h"

void linear(float *src, int srcRows, int srcCols,
              float *out, int outRows, int outCols) {
  
  float *w, *d_src, *d_out;
  
  // allocate device memory for output
  hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));

  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  
  // allocate device memory for weight
  hipMalloc((void**) &w, srcCols*outCols*sizeof(float));
  
  // init weight matrix
  initIdentityGPU<<<srcCols*outCols,1>>>(&w, srcCols, outCols);
  
  gpu_blas_mmul(w, d_src, d_out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);
  
  hipMemcpy(out,d_out,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  
}
