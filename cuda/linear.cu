#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <hipblas.h>
#include "cuBlasUtil.h"

void linear(float *src, int srcRows, int srcCols
              float *out, int outRows, int outCols,
               float *bias) {
  
  float *w;
  
  // allocate device memory for output
  //hipMalloc(&y,outRows*outCols*sizeof(float));
  
  // allocate device memory for weight
  hipMalloc(&w,srcCols*sizeof(float));
  
  // init weight matrix
  initIdentityGPU<<<srcCols/128,128>>>(&w, srcCols, 1);
  
  gpu_blas_mmul(w, src, out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);
  
  //hipMemcpy(out,y,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  
}
