#include "linear.h"

float* linear(float *src, int srcRows, int srcCols,
              float *out, int outRows, int outCols) {
  
  float *w, *d_src, *d_out;
  
  // allocate device memory for output
  hipError_t e = hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));

  const char* err = hipGetErrorString(e);

  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  e = hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  err = hipGetErrorString(e);  


  // allocate device memory for weight
  hipMalloc((void**) &w, srcCols*outCols*sizeof(float));

  float *h_w = (float*)calloc(srcCols*outCols, sizeof(float));
  memset(h_w, 1, srcCols*outCols*sizeof(float));
  hipMemcpy(w, h_w, srcCols*outCols*sizeof(float), hipMemcpyHostToDevice);


  // init weight matrix
  //initIdentityGPU<<<srcCols*outCols,1>>>(&w, srcCols, outCols);
  
  gpu_blas_mmul(w, d_src, d_out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);
  
  hipMemcpy(out,d_out,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  
  return out;
}
