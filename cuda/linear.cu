#include "linear.h"

void linear(float *src, int srcRows, int srcCols,
              float *out, int outRows, int outCols) {
  
  float *w, *d_src, *d_out;
  
  // allocate device memory for output
  hipMalloc(&d_out, outRows*outCols*sizeof(float));

  hipMalloc(&d_src, srcRows*srcCols*sizeof(float));
  hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  
  // allocate device memory for weight
  hipMalloc(&w, srcCols*outCols*sizeof(float));
  
  // init weight matrix
  initIdentityGPU<<<srcCols/128,128>>>(&w, srcCols, outCols);
  
  gpu_blas_mmul(w, d_src, d_out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);
  
  //hipMemcpy(out,y,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  
}
