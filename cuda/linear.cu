#include "linear.h"

void linear(float *src, int srcRows, int srcCols,
              float *out, int outRows, int outCols) {
  
  float *w, *d_src, *d_out;
  
  // allocate device memory for output
  hipMalloc((void**) &d_out, outRows*outCols*sizeof(float));

  hipMalloc((void**) &d_src, srcRows*srcCols*sizeof(float));
  hipMemcpy(d_src, src, srcRows*srcCols*sizeof(float), hipMemcpyHostToDevice);
  
  // allocate device memory for weight
  hipMalloc((void**) &w, srcCols*outCols*sizeof(float));

  float *h_w = (float*)calloc(srcCols*outCols, sizeof(float));
  memset(h_w, 1, srcCols*outCols*sizeof(float));
  hipMemcpy(w, h_w, srcCols*outCols*sizeof(float), hipMemcpyHostToDevice);


  // init weight matrix
  //initIdentityGPU<<<srcCols*outCols,1>>>(&w, srcCols, outCols);
  
  gpu_blas_mmul(w, d_src, d_out, srcRows, srcCols, outCols, false, false, 1.0, 0.0);
  
  hipMemcpy(out,d_out,outRows*outCols*sizeof(float),hipMemcpyDeviceToHost);
  
}
