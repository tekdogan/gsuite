#include"DataLoader.h"

#define SIZE 1024

#ifdef __cplusplus
extern "C" {
#endif

int LoadData() {

	float *h_edgeIndex, *h_featureVector;
	float *edgeIndex, *featureVector;

	float h_aggregationVar[2] = {0,0};
	float h_nodeDegrees[3] = {1,2,1};

	float aggregationVar[2], nodeDegrees[3];

	std::unordered_map<int,int> nodeMap;

	hipMallocManaged( (void**) &aggregationVar, 2 * sizeof(float));
	hipMemcpy(aggregationVar, h_aggregationVar, 2*sizeof(float), hipMemcpyHostToDevice);

	hipMallocManaged( (void**) &nodeDegrees, 3*sizeof(float));
	hipMemcpy(nodeDegrees, h_nodeDegrees, 3*sizeof(float), hipMemcpyHostToDevice);

	const char* edgeIndexFileName = "cora.cites.bak2";
	int edgeIndexSize = getEdgeIndexSizeFromFile(edgeIndexFileName);
	std::cout << "edgeIndexSize: " << edgeIndexSize << std::endl;

	const char* featureFileName = "cora.content.bak2";
	int featureSize = getFeatureSizeFromFile(featureFileName);
	std::cout << "featureSize: " << featureSize << std::endl;

	int numOfNodes = getNumOfNodesFromFile(featureFileName);
	std::cout << "numOfNodes: " << numOfNodes << std::endl;

	try {
		h_featureVector = (float*) calloc(numOfNodes*featureSize, sizeof(float));
		loadFeatureVectorFromFile(featureFileName, h_featureVector, featureSize, nodeMap);
		hipMallocManaged( (void**) &featureVector, numOfNodes*featureSize * sizeof(float));
		hipMemcpy(featureVector, h_featureVector, numOfNodes*featureSize * sizeof(float), hipMemcpyHostToDevice);
	} catch(...) {
		std::cout << "Could not allocate memory space for featureVector!\n";
	}
	
	try {
		h_edgeIndex = (float*) calloc(featureSize*edgeIndexSize, sizeof(float));
		loadEdgeIndexFromFile(edgeIndexFileName, h_edgeIndex, edgeIndexSize, nodeMap);
		hipMallocManaged( (void**) &edgeIndex, featureSize*edgeIndexSize * sizeof(float));
		hipMemcpy(edgeIndex, h_edgeIndex, (size_t)featureSize*edgeIndexSize*sizeof(float), hipMemcpyHostToDevice);
		
	} catch(...) {
		std::cout << "Could not allocate memory space for edgeIndex!\n";
	}

	auto start = std::chrono::steady_clock::now();

	//CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees);

	//CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees);

	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "2-layer GCN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
	//	std::cout << "Node" << i << " feature 0: " << *(featureVector + featureSize*i) << std::endl;
	//	std::cout << "Node" << i << " feature 1: " << *(featureVector + featureSize*i + 1) << std::endl;
	//}


	float *featureVectorOutput;
	hipMallocManaged( (void**) &featureVectorOutput, numOfNodes*featureSize * sizeof(float));
        start = std::chrono::steady_clock::now();
        //GINLayer<<<1,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, 1.0, featureVectorOutput);
        end = std::chrono::steady_clock::now();
        dur_ms = end-start;
        std::cout << "1-layer GIN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
        //        std::cout << "Node" << i << " feature 0: " << *(featureVectorOutput + featureSize*i) << std::endl;
        //        std::cout << "Node" << i << " feature 1: " << *(featureVectorOutput + featureSize*i + 1) << std::endl;
        //}

	//float *adjMatrix = (float*)calloc(numOfNodes*numOfNodes, sizeof(float));
	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));

	std::cout << "DEBUG: coo2sparse operation start...\n";
        coo2sparse(edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	/*for(int i=0; i<numOfNodes; i++) {
                for(int j=0; j<numOfNodes; j++) {
                        std::cout << *(adjMatrix + i*numOfNodes + j) << " ";
                }
                std::cout << std::endl;
        }*/

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

	std::cout << "DEBUG: CU_SpMM::GCN start...\n";
	start = std::chrono::steady_clock::now();
	CU_SpMM::GCNLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix);
	end = std::chrono::steady_clock::now();
        dur_ms = end-start;
        std::cout << "1-layer CU_SpMM::GCN execution took " << dur_ms.count() << " ms\n";

	std::cout << "CU_SpMM::GCNLayer operation returned successfully!\n";

	//for(int i=0; i<featureSize; i++) {
	//	for(int j=0; j<numOfNodes; j++) {
	//		std::cout << *(outputMatrix + i*numOfNodes + j) << " ";
	//	}
	//	std::cout << std::endl;
	//}

	hipFree(edgeIndex);
	hipFree(featureVector);
	hipFree(featureVectorOutput);
	hipFree(aggregationVar);
	hipFree(nodeDegrees);

	//std::cout << "nodeMap.size(): " << nodeMap.size() << std::endl;
	//for( const std::pair<int,int>& n : nodeMap ) {
	//	std::cout << "Key:[" << n.first << "] Value:[" << n.second << "]\n";
	//}

	return 0;
}

int getEdgeIndexSizeFromFile(const char* fileName) {

	std::ifstream dsFile(fileName);
	std::string line;

	int numOfEdges = 0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			numOfEdges++;
		}
	}
	else {
		std::cout << "Could not open the edgeIndex dataset file!\n";
		return -1;
	}

	return numOfEdges;

}

void loadEdgeIndexFromFile(const char* fileName, float* edgeIndex, const int numOfEdges,
			   std::unordered_map<int, int> &nodeMap) {

	std::ifstream dsFile(fileName);
	std::string line;

	int i=0, j=0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			std::istringstream ss(line);
			std::string word;
			while(std::getline(ss, word, '\t')) {
				*(edgeIndex + (numOfEdges*i) + j) = nodeMap.find(std::stof(word))->second;
				i = 1;
			}
		j++;
		i = 0;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
	}

}

int getFeatureSizeFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
	std::string line, word;

	int numOfFeatures = -1;

	if(dsFile.is_open()) {
		getline(dsFile, line);
		std::istringstream ss(line);
		while(std::getline(ss, word, '\t')) {
			numOfFeatures++;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
		return -1;
	}

	return numOfFeatures;
}

int getNumOfNodesFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
        std::string line;

        int numOfNodes = 0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
			numOfNodes++;
		}
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
                return -1;
        }

        return numOfNodes;

}

void loadFeatureVectorFromFile(const char* fileName, float* featureVector, int featureSize, std::unordered_map<int, int> &nodeMap) {
	std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
                        std::istringstream ss(line);
                        std::string word;
			std::getline(ss, word, '\t'); // escape node index
			nodeMap.insert({std::stof(word),i});
                        while(std::getline(ss, word, '\t')) {
				if(word.length() < 5) {
                                    *(featureVector + i*featureSize + j) = std::stof(word);
                                    j += 1;
				}
                        }
                i++;
                j = 0;
                }
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
        }
}


#ifdef __cplusplus
}
#endif
