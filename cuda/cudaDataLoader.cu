#include"DataLoader.h"

// number of threads per block
#define TPB 1
#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

#include "CU_SpMM_GCN.h"
#include "CU_SpMM_GIN.h"
#include "C_GCN_MP.h"
#include "CU_GIN_WL.h"
#include "CU_SAG_MP.h"

#ifdef __cplusplus
extern "C" {
#endif

int LoadData(int arg) {


	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );
	float *h_edgeIndex, *h_featureVector;

	std::unordered_map<std::string,std::string> nodeMap;

	const char* edgeIndexFileName = "pubmed.cites";
	int edgeIndexSize = getEdgeIndexSizeFromFile(edgeIndexFileName);
	std::cout << "edgeIndexSize: " << edgeIndexSize << std::endl;

	const char* featureFileName = "pubmed.content";
	int featureSize = getFeatureSizeFromFile(featureFileName);
	std::cout << "featureSize: " << featureSize << std::endl;

	int numOfNodes = getNumOfNodesFromFile(featureFileName);
	std::cout << "numOfNodes: " << numOfNodes << std::endl;

	try {
		h_featureVector = (float*) calloc(numOfNodes*featureSize, sizeof(float));
		loadFeatureVectorFromFile(featureFileName, h_featureVector, featureSize, nodeMap);
	} catch(...) {
		std::cout << "Could not allocate memory space for featureVector!\n";
	}
	
	std::cout << "nodeMap.size(): " << nodeMap.size() << std::endl;

	try {
		h_edgeIndex = (float*) calloc(2*edgeIndexSize, sizeof(float));
		loadEdgeIndexFromFile(edgeIndexFileName, h_edgeIndex, edgeIndexSize, nodeMap);
	} catch(...) {
		std::cout << "Could not allocate memory space for edgeIndex!\n";
	}

	int *h_edgeIndexInt = (int*)calloc(2*edgeIndexSize, sizeof(int));
	for(int i=0; i<2*edgeIndexSize; i++) {
		*(h_edgeIndexInt + i) = (int)*(h_edgeIndex + i);
	}

	int *arr = (int*)calloc(2*edgeIndexSize, sizeof(int));

	if(arg == 0) { // execute CU_MP_GCN

	auto start = std::chrono::steady_clock::now();

        for(int i = 0; i<1; i++) {
		hipProfilerStart();
		float* o = CU_MP::GCNLayer(h_edgeIndexInt, h_featureVector, numOfNodes, featureSize, edgeIndexSize, 1028);
		//o = CU_MP::GCNLayer(h_edgeIndexInt, o, numOfNodes, 16, edgeIndexSize, 7);	
		free(o);
		hipProfilerStop();
	}


	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "2-layer GCN execution took " << dur_ms.count() << " ms\n";


	} // CU_MP_GCN end

	else if(arg == 1) { // execute CU_SpMM::GIN

	float *featureVectorOutput;
	hipMalloc( (void**) &featureVectorOutput, numOfNodes*featureSize * sizeof(float));

	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));
	std::cout << "DEBUG: coo2sparse operation start...\n";
	coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

        auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
        CU_SpMM::GINLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix, 0.1);
	hipProfilerStop();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer GIN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
        //        std::cout << "Node" << i << " feature 0: " << *(featureVectorOutput + featureSize*i) << std::endl;
        //        std::cout << "Node" << i << " feature 1: " << *(featureVectorOutput + featureSize*i + 1) << std::endl;
        //}

	} // CU_SpMM::GIN end

	else if(arg == 2) { // execute CU_SpMM_GCN

	//float *adjMatrix = (float*)calloc(numOfNodes*numOfNodes, sizeof(float));
	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));

	std::cout << "DEBUG: coo2sparse operation start...\n";
        coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	/*for(int i=0; i<numOfNodes; i++) {
                for(int j=0; j<numOfNodes; j++) {
                        std::cout << *(adjMatrix + i*numOfNodes + j) << " ";
                }
                std::cout << std::endl;
        }*/

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

	std::cout << "DEBUG: CU_SpMM::GCN start...\n";
	auto start = std::chrono::steady_clock::now();
	// hipProfilerStart();
	CU_SpMM::GCNLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix);
	// hipProfilerStop();
	auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer CU_SpMM::GCN execution took " << dur_ms.count() << " ms\n";

	std::cout << "CU_SpMM::GCNLayer operation returned successfully!\n";

	//for(int i=0; i<featureSize; i++) {
	//	for(int j=0; j<numOfNodes; j++) {
	//		std::cout << *(outputMatrix + i*numOfNodes + j) << " ";
	//	}
	//	std::cout << std::endl;
	//}

	} // CU_SpMM_GCN end

	else if(arg == 3) { // execute CU_MP::SAG
		hipProfilerStart();
		float* output = CU_MP::SAGELayer(h_edgeIndexInt, h_featureVector, numOfNodes, featureSize, edgeIndexSize, 1024);
		hipProfilerStop();
                free(output);
	}
	else if(arg == 4) { // execute CU_WL::GIN

		float* output = CU_WL::GINLayer(h_edgeIndexInt, h_featureVector, numOfNodes, featureSize, edgeIndexSize, 1024, 0.01);
		free(output);		
	}

	//std::cout << "edge index matrix before sort:\n";
	//printDenseMatrix(h_edgeIndex, 2, edgeIndexSize);
	//std::cout << std::endl;

	//for(int i=0; i<edgeIndexSize*2; i++)
	//	std::cout << "#" << *(h_edgeIndex+i);
	//std::cout << std::endl;

	//std::cout << "feature matrix:\n";
	//printDenseMatrix(h_featureVector, featureSize, numOfNodes);
	//std::cout << std::endl;

	// hipFree(edgeIndex);
	// hipFree(featureVector);
	//hipFree(featureVectorOutput);
	// hipFree(aggregationVar);
	// hipFree(nodeDegrees);

	//std::cout << "nodeMap.size(): " << nodeMap.size() << std::endl;
	//for( const std::pair<int,int>& n : nodeMap ) {
	//	std::cout << "Key:[" << n.first << "] Value:[" << n.second << "]\n";
	//}

	/*int *arr = (int*)calloc(2*edgeIndexSize, sizeof(int));

	for(int i=0; i<8; i++) {
			//std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
			*(arr + i) = (int)( *(h_edgeIndex + i));
	}

	gpu_qsort(arr, 4, edgeIndexSize);

	for(int i=0; i<8; i++) {
	                //std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
	                ( *(h_edgeIndex + i)) = (float)( *(arr + i));
	}*/

	//std::cout << "edge index matrix after sort:\n";
	//printDenseMatrix(h_edgeIndex, 2, edgeIndexSize);
	//std::cout << std::endl;
	

	return 0;
}

int getEdgeIndexSizeFromFile(const char* fileName) {

	std::ifstream dsFile(fileName);
	std::string line;

	int numOfEdges = 0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			numOfEdges++;
		}
	}
	else {
		std::cout << "Could not open the edgeIndex dataset file!\n";
		return -1;
	}

	return numOfEdges;

}

void loadEdgeIndexFromFile(const char* fileName, float* edgeIndex, const int numOfEdges,
			   std::unordered_map<std::string, std::string> &nodeMap) {

	std::ifstream dsFile(fileName);
	std::string line;

	std::cout << fileName << std::endl;

	char sep = '\t';

	if( strcmp( fileName,"cora.cites.bak2") == 0 ) {
		std::cout << "DEBUG: CORA dataset edges are loading..." << std::endl;
		sep = '\t';
	}
	if( strcmp( fileName,"citeseer.cites") == 0 ) {
		std::cout << "DEBUG: CiteSeer dataset edges are loading..." << std::endl;
		sep = '\t';
	}
	if( strcmp( fileName,"pubmed.cites" ) == 0 ) {
		std::cout << "DEBUG: PubMed dataset edges are loading..." << std::endl;
		sep = ',';

	}

	int i=0, j=0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			std::istringstream ss(line);
			std::string word;
			while(std::getline(ss, word, sep)) {
				*(edgeIndex + (numOfEdges*i) + j) = std::stoi( nodeMap.find(word)->second );
				i = 1;
			}
		j++;
		i = 0;
		}
		std::cout << "DEBUG: edgeIndex loaded! j=" << j << std::endl;
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
	}

}



int getFeatureSizeFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
	std::string line, word;

	int numOfFeatures = -1;

	if( strcmp(fileName,"pubmed.content") == 0 )
		return 500;

	if(dsFile.is_open()) {
		getline(dsFile, line);
		std::istringstream ss(line);
		while(std::getline(ss, word, '\t')) {
			numOfFeatures++;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
		return -1;
	}

	return numOfFeatures;
}

int getNumOfNodesFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
        std::string line;

        int numOfNodes = 0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
			numOfNodes++;
		}
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
                return -1;
        }

        return numOfNodes;

}

void loadFeatureVectorFromFile(const char* fileName, float* featureVector, int featureSize, std::unordered_map<std::string, std::string> &nodeMap) {
	std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

	if( strcmp(fileName,"cora.content.bak2") == 0 )
	{
		std::cout << "DEBUG: CORA dataset features are loading..." << std::endl;

	        if(dsFile.is_open()) {
	                while(getline(dsFile, line)) {
	                        std::istringstream ss(line);
	                        std::string word;
				std::getline(ss, word, '\t'); // escape node index
				nodeMap.insert({word,std::to_string(i)});
	                        while(std::getline(ss, word, '\t')) {
					if(word.length() < 5) {
	                                    *(featureVector + i*featureSize + j) = std::stof(word);
	                                    j += 1;
					}
	                        }
	                i++;
	                j = 0;
	                }
	        }
	        else {
	                std::cout << "Could not open the feature dataset file!\n";
	        }


	}
	else if( strcmp(fileName,"citeseer.content") == 0 )
	{

		std::cout << "DEBUG: CiteSeer dataset features are loading...\n";

                if(dsFile.is_open()) {
                        while(getline(dsFile, line)) {
                                std::istringstream ss(line);
                                std::string word;
                                std::getline(ss, word, '\t'); // escape node index
                                nodeMap.insert({word,std::to_string(i)});
                                while(std::getline(ss, word, '\t')) {
                                        if(word.length() < 2) {
                                            *(featureVector + i*featureSize + j) = std::stof(word);
                                            j += 1;
                                        }
                                }
                        i++;
                        j = 0;
                        }
                }
                else {
                        std::cout << "Could not open the feature dataset file!\n";
                }

	}
	else if( strcmp(fileName,"pubmed.content") == 0 )
        {

                std::cout << "DEBUG: PubMed dataset features are loading...\n";

                if(dsFile.is_open()) {
                        while(getline(dsFile, line)) {
                                std::istringstream ss(line);
                                std::string word;
                                std::getline(ss, word, ','); // escape node index
                                nodeMap.insert({word,std::to_string(i)});
                                while(std::getline(ss, word, ',')) {
                                        while(j<500) {
                                            *(featureVector + i*featureSize + j) = std::stof(word);
                                            j += 1;
                                        }
                                }
                        i++;
                        j = 0;
                        }
                }
                else {
                        std::cout << "Could not open the feature dataset file!\n";
                }

        }

}


#ifdef __cplusplus
}
#endif
