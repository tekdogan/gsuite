#include"DataLoader.h"

// number of threads per block
#define TPB 512

#ifdef __cplusplus
extern "C" {
#endif

int LoadData(int arg) {


	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );
	float *h_edgeIndex, *h_featureVector;
	float *edgeIndex, *featureVector;

	std::unordered_map<int,int> nodeMap;

	const char* edgeIndexFileName = "cora.cites.bak2";
	int edgeIndexSize = getEdgeIndexSizeFromFile(edgeIndexFileName);
	std::cout << "edgeIndexSize: " << edgeIndexSize << std::endl;

	const char* featureFileName = "cora.content.bak2";
	int featureSize = getFeatureSizeFromFile(featureFileName);
	std::cout << "featureSize: " << featureSize << std::endl;

	int numOfNodes = getNumOfNodesFromFile(featureFileName);
	std::cout << "numOfNodes: " << numOfNodes << std::endl;

	try {
		h_featureVector = (float*) calloc(numOfNodes*featureSize, sizeof(float));
		loadFeatureVectorFromFile(featureFileName, h_featureVector, featureSize, nodeMap);
		hipMalloc( (void**) &featureVector, numOfNodes*featureSize * sizeof(float));
		hipMemcpy(featureVector, h_featureVector, numOfNodes*featureSize * sizeof(float), hipMemcpyHostToDevice);
	} catch(...) {
		std::cout << "Could not allocate memory space for featureVector!\n";
	}
	
	try {
		h_edgeIndex = (float*) calloc(2*edgeIndexSize, sizeof(float));
		loadEdgeIndexFromFile(edgeIndexFileName, h_edgeIndex, edgeIndexSize, nodeMap);
		hipMalloc( (void**) &edgeIndex, 2*edgeIndexSize * sizeof(float));
		hipMemcpy(edgeIndex, h_edgeIndex, (size_t)2*edgeIndexSize*sizeof(float), hipMemcpyHostToDevice);
		
	} catch(...) {
		std::cout << "Could not allocate memory space for edgeIndex!\n";
	}

        float *h_aggregationVar = (float*)calloc(numOfNodes * featureSize, sizeof(float));
	float *h_nodeDegrees = (float*)calloc(numOfNodes, sizeof(float));
        float *aggregationVar, *nodeDegrees;

	for(int i=0; i<edgeIndexSize; i++) {
		h_nodeDegrees[(int)(*(h_edgeIndex + i))]++;
	}

	for(int i=0; i<numOfNodes*featureSize; i++) {
		*(h_aggregationVar + i) = 0.0;
	}

	hipMalloc( (void**) &aggregationVar, numOfNodes * featureSize * sizeof(float));
	hipMemcpy(aggregationVar, h_aggregationVar, numOfNodes * featureSize * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc( (void**) &nodeDegrees, featureSize*sizeof(float));
	hipMemcpy(nodeDegrees, h_nodeDegrees, featureSize*sizeof(float), hipMemcpyHostToDevice);

	if(arg == 0) { // execute CU_MP_GCN

	auto start = std::chrono::steady_clock::now();

	hipProfilerStart();

	CU_MP::GCNLayerNew<<<numOfNodes/TPB,TPB>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, numOfNodes, featureSize, edgeIndexSize);

	//CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, numOfNodes, featureSize, edgeIndexSize);

	hipProfilerStop();

	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "2-layer GCN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
	//	std::cout << "Node" << i << " feature 0: " << *(featureVector + featureSize*i) << std::endl;
	//	std::cout << "Node" << i << " feature 1: " << *(featureVector + featureSize*i + 1) << std::endl;
	//}

	} // CU_MP_GCN end

	else if(arg == 1) { // execute CU_SpMM::GIN

	float *featureVectorOutput;
	hipMalloc( (void**) &featureVectorOutput, numOfNodes*featureSize * sizeof(float));

	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));
	std::cout << "DEBUG: coo2sparse operation start...\n";
	coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

        auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
        CU_SpMM::GINLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix, 0.1);
	hipProfilerStop();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer GIN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
        //        std::cout << "Node" << i << " feature 0: " << *(featureVectorOutput + featureSize*i) << std::endl;
        //        std::cout << "Node" << i << " feature 1: " << *(featureVectorOutput + featureSize*i + 1) << std::endl;
        //}

	} // CU_SpMM::GIN end

	else if(arg == 2) { // execute CU_SpMM_GCN

	//float *adjMatrix = (float*)calloc(numOfNodes*numOfNodes, sizeof(float));
	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));

	std::cout << "DEBUG: coo2sparse operation start...\n";
        coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	/*for(int i=0; i<numOfNodes; i++) {
                for(int j=0; j<numOfNodes; j++) {
                        std::cout << *(adjMatrix + i*numOfNodes + j) << " ";
                }
                std::cout << std::endl;
        }*/

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

	std::cout << "DEBUG: CU_SpMM::GCN start...\n";
	auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
	CU_SpMM::GCNLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix);
	hipProfilerStop();
	auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer CU_SpMM::GCN execution took " << dur_ms.count() << " ms\n";

	std::cout << "CU_SpMM::GCNLayer operation returned successfully!\n";

	//for(int i=0; i<featureSize; i++) {
	//	for(int j=0; j<numOfNodes; j++) {
	//		std::cout << *(outputMatrix + i*numOfNodes + j) << " ";
	//	}
	//	std::cout << std::endl;
	//}

	} // CU_SpMM_GCN end

	else if(arg == 3) { // execute CU_WL::SAG

	float* outputFeatureMatrix, *tempFeatureValues;
	hipMalloc(&outputFeatureMatrix, featureSize * numOfNodes * sizeof(float));	
	hipMalloc(&tempFeatureValues, featureSize * numOfNodes * sizeof(float));

	auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
	CU_WL::SAGLayer<<<numOfNodes/TPB,TPB>>>(edgeIndex, featureVector, 1.0, 0.2, numOfNodes, edgeIndexSize, featureSize, tempFeatureValues, outputFeatureMatrix);
	hipProfilerStop();
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "1-layer CU_WL::SAG execution took " << dur_ms.count() << " ms\n";

	float* h_outputFeatureMatrix = (float*)calloc(featureSize * numOfNodes, sizeof(float));
	hipMemcpy(h_outputFeatureMatrix, outputFeatureMatrix, featureSize * numOfNodes * sizeof(float), hipMemcpyDeviceToHost);

	//std::cout << "Output feature matrix:\n";
	//printDenseMatrix(h_outputFeatureMatrix, featureSize, numOfNodes);

	} // CU_WL::SAG end

	else if(arg == 4) { // execute CU_WL::GIN

        float* outputFeatureMatrix, *tempFeatureValues;
        hipMalloc(&outputFeatureMatrix, featureSize * numOfNodes * sizeof(float));
        hipMalloc(&tempFeatureValues, featureSize * numOfNodes * sizeof(float));

        auto start = std::chrono::steady_clock::now();
        hipProfilerStart();
        CU_WL::GINLayer<<<numOfNodes/TPB,TPB>>>(edgeIndex, featureVector, tempFeatureValues, 0.3, numOfNodes, edgeIndexSize, featureSize, outputFeatureMatrix);
        hipProfilerStop();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer CU_WL::GIN execution took " << dur_ms.count() << " ms\n";

        float* h_outputFeatureMatrix = (float*)calloc(featureSize * numOfNodes, sizeof(float));
        hipMemcpy(h_outputFeatureMatrix, outputFeatureMatrix, featureSize * numOfNodes * sizeof(float), hipMemcpyDeviceToHost);

	}

	//std::cout << "edge index matrix:\n";
	//printDenseMatrix(h_edgeIndex, edgeIndexSize, 2);
	//std::cout << std::endl;

	//std::cout << "feature matrix:\n";
	//printDenseMatrix(h_featureVector, featureSize, numOfNodes);
	//std::cout << std::endl;

	hipFree(edgeIndex);
	hipFree(featureVector);
	//hipFree(featureVectorOutput);
	hipFree(aggregationVar);
	hipFree(nodeDegrees);

	//std::cout << "nodeMap.size(): " << nodeMap.size() << std::endl;
	//for( const std::pair<int,int>& n : nodeMap ) {
	//	std::cout << "Key:[" << n.first << "] Value:[" << n.second << "]\n";
	//}

	return 0;
}

int getEdgeIndexSizeFromFile(const char* fileName) {

	std::ifstream dsFile(fileName);
	std::string line;

	int numOfEdges = 0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			numOfEdges++;
		}
	}
	else {
		std::cout << "Could not open the edgeIndex dataset file!\n";
		return -1;
	}

	return numOfEdges;

}

void loadEdgeIndexFromFile(const char* fileName, float* edgeIndex, const int numOfEdges,
			   std::unordered_map<int, int> &nodeMap) {

	std::ifstream dsFile(fileName);
	std::string line;

	int i=0, j=0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			std::istringstream ss(line);
			std::string word;
			while(std::getline(ss, word, '\t')) {
				*(edgeIndex + (numOfEdges*i) + j) = nodeMap.find(std::stof(word))->second;
				i = 1;
			}
		j++;
		i = 0;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
	}

}

int getFeatureSizeFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
	std::string line, word;

	int numOfFeatures = -1;

	if(dsFile.is_open()) {
		getline(dsFile, line);
		std::istringstream ss(line);
		while(std::getline(ss, word, '\t')) {
			numOfFeatures++;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
		return -1;
	}

	return numOfFeatures;
}

int getNumOfNodesFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
        std::string line;

        int numOfNodes = 0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
			numOfNodes++;
		}
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
                return -1;
        }

        return numOfNodes;

}

void loadFeatureVectorFromFile(const char* fileName, float* featureVector, int featureSize, std::unordered_map<int, int> &nodeMap) {
	std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
                        std::istringstream ss(line);
                        std::string word;
			std::getline(ss, word, '\t'); // escape node index
			nodeMap.insert({std::stof(word),i});
                        while(std::getline(ss, word, '\t')) {
				if(word.length() < 5) {
                                    *(featureVector + i*featureSize + j) = std::stof(word);
                                    j += 1;
				}
                        }
                i++;
                j = 0;
                }
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
        }
}


#ifdef __cplusplus
}
#endif
