#include"DataLoader.h"

// number of threads per block
#define TPB 1
#define NUM_OF_THREADS 1024

#ifdef __cplusplus
extern "C" {
#endif

int LoadData(int arg) {


	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );
	float *h_edgeIndex, *h_featureVector;
	float *edgeIndex, *featureVector;

	std::unordered_map<int,int> nodeMap;

	const char* edgeIndexFileName = "cora.cites";
	int edgeIndexSize = getEdgeIndexSizeFromFile(edgeIndexFileName);
	std::cout << "edgeIndexSize: " << edgeIndexSize << std::endl;

	const char* featureFileName = "cora.content";
	int featureSize = getFeatureSizeFromFile(featureFileName);
	std::cout << "featureSize: " << featureSize << std::endl;

	int numOfNodes = getNumOfNodesFromFile(featureFileName);
	std::cout << "numOfNodes: " << numOfNodes << std::endl;

	try {
		h_featureVector = (float*) calloc(numOfNodes*featureSize, sizeof(float));
		loadFeatureVectorFromFile(featureFileName, h_featureVector, featureSize, nodeMap);
		hipMalloc( (void**) &featureVector, numOfNodes*featureSize * sizeof(float));
		hipMemcpy(featureVector, h_featureVector, numOfNodes*featureSize * sizeof(float), hipMemcpyHostToDevice);
	} catch(...) {
		std::cout << "Could not allocate memory space for featureVector!\n";
	}
	
	try {
		h_edgeIndex = (float*) calloc(2*edgeIndexSize, sizeof(float));
		loadEdgeIndexFromFile(edgeIndexFileName, h_edgeIndex, edgeIndexSize, nodeMap);
		hipMalloc( (void**) &edgeIndex, 2*edgeIndexSize * sizeof(float));
		hipMemcpy(edgeIndex, h_edgeIndex, (size_t)2*edgeIndexSize*sizeof(float), hipMemcpyHostToDevice);
		
	} catch(...) {
		std::cout << "Could not allocate memory space for edgeIndex!\n";
	}

        float *h_aggregationVar = (float*)calloc(numOfNodes * featureSize, sizeof(float));
	float *h_nodeDegrees = (float*)calloc(numOfNodes, sizeof(float));
        float *aggregationVar, *nodeDegrees;

	for(int i=0; i<edgeIndexSize; i++) {
		h_nodeDegrees[(int)(*(h_edgeIndex + i))]++;
	}

	for(int i=0; i<numOfNodes*featureSize; i++) {
		*(h_aggregationVar + i) = 0.0;
	}

	hipMalloc( (void**) &aggregationVar, numOfNodes * featureSize * sizeof(float));
	hipMemcpy(aggregationVar, h_aggregationVar, numOfNodes * featureSize * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc( (void**) &nodeDegrees, featureSize*sizeof(float));
	hipMemcpy(nodeDegrees, h_nodeDegrees, featureSize*sizeof(float), hipMemcpyHostToDevice);


	int *arr = (int*)calloc(2*edgeIndexSize, sizeof(int));

	/*for(int i=0; i<8; i++) {
	                //std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
	                *(arr + i) = (int)( *(h_edgeIndex + i));
	}

	gpu_qsort(arr, edgeIndexSize, edgeIndexSize);

	for(int i=0; i<8; i++) {
	                //std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
	                ( *(h_edgeIndex + i)) = (float)( *(arr + i));
	}*/



	//unordered_map<string, double>:: iterator itr;
	//for (itr = umap.begin(); itr != umap.end(); itr++)

	if(arg == 0) { // execute CU_MP_GCN

	auto start = std::chrono::steady_clock::now();

	hipProfilerStart();

	CU_MP::GCNLayerNew<<<numOfNodes,512>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, numOfNodes, featureSize, edgeIndexSize);

	//CU_MP::GCNLayerNew<<<16,SIZE>>>(edgeIndex, featureVector, aggregationVar, nodeDegrees, numOfNodes, featureSize, edgeIndexSize);

	hipProfilerStop();

	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "2-layer GCN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
	//	std::cout << "Node" << i << " feature 0: " << *(featureVector + featureSize*i) << std::endl;
	//	std::cout << "Node" << i << " feature 1: " << *(featureVector + featureSize*i + 1) << std::endl;
	//}

	} // CU_MP_GCN end

	else if(arg == 1) { // execute CU_SpMM::GIN

	float *featureVectorOutput;
	hipMalloc( (void**) &featureVectorOutput, numOfNodes*featureSize * sizeof(float));

	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));
	std::cout << "DEBUG: coo2sparse operation start...\n";
	coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

        auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
        CU_SpMM::GINLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix, 0.1);
	hipProfilerStop();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer GIN execution took " << dur_ms.count() << " ms\n";

	//for(int i=0; i<3; i++) {
        //        std::cout << "Node" << i << " feature 0: " << *(featureVectorOutput + featureSize*i) << std::endl;
        //        std::cout << "Node" << i << " feature 1: " << *(featureVectorOutput + featureSize*i + 1) << std::endl;
        //}

	} // CU_SpMM::GIN end

	else if(arg == 2) { // execute CU_SpMM_GCN

	//float *adjMatrix = (float*)calloc(numOfNodes*numOfNodes, sizeof(float));
	float *adjMatrix = (float*)calloc(edgeIndexSize*edgeIndexSize, sizeof(float));

	std::cout << "DEBUG: coo2sparse operation start...\n";
        coo2sparse(h_edgeIndex, adjMatrix, edgeIndexSize, numOfNodes);
	std::cout << "DEBUG: coo2sparse operation successful!\n";

	/*for(int i=0; i<numOfNodes; i++) {
                for(int j=0; j<numOfNodes; j++) {
                        std::cout << *(adjMatrix + i*numOfNodes + j) << " ";
                }
                std::cout << std::endl;
        }*/

	float* outputMatrix = (float*)calloc(numOfNodes * featureSize, sizeof(float));

	std::cout << "DEBUG: CU_SpMM::GCN start...\n";
	auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
	CU_SpMM::GCNLayer(adjMatrix, h_featureVector, numOfNodes, edgeIndexSize, featureSize, outputMatrix);
	hipProfilerStop();
	auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer CU_SpMM::GCN execution took " << dur_ms.count() << " ms\n";

	std::cout << "CU_SpMM::GCNLayer operation returned successfully!\n";

	//for(int i=0; i<featureSize; i++) {
	//	for(int j=0; j<numOfNodes; j++) {
	//		std::cout << *(outputMatrix + i*numOfNodes + j) << " ";
	//	}
	//	std::cout << std::endl;
	//}

	} // CU_SpMM_GCN end

	else if(arg == 3) { // execute CU_WL::SAG

	float* outputFeatureMatrix, *tempFeatureValues;
	hipMalloc(&outputFeatureMatrix, featureSize * numOfNodes * sizeof(float));	
	hipMalloc(&tempFeatureValues, featureSize * numOfNodes * sizeof(float));

	auto start = std::chrono::steady_clock::now();
	hipProfilerStart();
	//CU_WL::SAGLayer<<<numOfNodes,512>>>(edgeIndex, featureVector, 1.0, 0.2, numOfNodes, edgeIndexSize, featureSize, tempFeatureValues, outputFeatureMatrix);
	//CU_WL::SAGLayer2<<<numOfNodes/TPB,TPB>>>(edgeIndex, featureVector, 1.0, 0.2, numOfNodes, edgeIndexSize, featureSize, tempFeatureValues, outputFeatureMatrix);
	hipProfilerStop();
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double, std::milli> dur_ms = end-start;
	std::cout << "1-layer CU_WL::SAG execution took " << dur_ms.count() << " ms\n";

	float* h_outputFeatureMatrix = (float*)calloc(featureSize * numOfNodes, sizeof(float));
	hipMemcpy(h_outputFeatureMatrix, outputFeatureMatrix, featureSize * numOfNodes * sizeof(float), hipMemcpyDeviceToHost);

	//std::cout << "Output feature matrix:\n";
	//printDenseMatrix(h_outputFeatureMatrix, featureSize, numOfNodes);

	} // CU_WL::SAG end

	else if(arg == 4) { // execute CU_WL::GIN

        float* outputFeatureMatrix, *tempFeatureValues;
        hipMalloc(&outputFeatureMatrix, featureSize * numOfNodes * sizeof(float));
        hipMalloc(&tempFeatureValues, featureSize * numOfNodes * sizeof(float));

        auto start = std::chrono::steady_clock::now();
        hipProfilerStart();
        CU_WL::GINLayer<<<numOfNodes*featureSize*edgeIndexSize,NUM_OF_THREADS>>>(edgeIndex, featureVector, tempFeatureValues, 0.3, numOfNodes, edgeIndexSize, featureSize, outputFeatureMatrix);
        hipProfilerStop();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur_ms = end-start;
        std::cout << "1-layer CU_WL::GIN execution took " << dur_ms.count() << " ms\n";

        float* h_outputFeatureMatrix = (float*)calloc(featureSize * numOfNodes, sizeof(float));
        hipMemcpy(h_outputFeatureMatrix, outputFeatureMatrix, featureSize * numOfNodes * sizeof(float), hipMemcpyDeviceToHost);

	}

	//std::cout << "edge index matrix before sort:\n";
	//printDenseMatrix(h_edgeIndex, 2, edgeIndexSize);
	//std::cout << std::endl;

	//for(int i=0; i<edgeIndexSize*2; i++)
	//	std::cout << "#" << *(h_edgeIndex+i);
	//std::cout << std::endl;

	//std::cout << "feature matrix:\n";
	//printDenseMatrix(h_featureVector, featureSize, numOfNodes);
	//std::cout << std::endl;

	hipFree(edgeIndex);
	hipFree(featureVector);
	//hipFree(featureVectorOutput);
	hipFree(aggregationVar);
	hipFree(nodeDegrees);

	//std::cout << "nodeMap.size(): " << nodeMap.size() << std::endl;
	//for( const std::pair<int,int>& n : nodeMap ) {
	//	std::cout << "Key:[" << n.first << "] Value:[" << n.second << "]\n";
	//}

	/*int *arr = (int*)calloc(2*edgeIndexSize, sizeof(int));

	for(int i=0; i<8; i++) {
			//std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
			*(arr + i) = (int)( *(h_edgeIndex + i));
	}

	gpu_qsort(arr, 4, edgeIndexSize);

	for(int i=0; i<8; i++) {
	                //std::cout << "copying: " << (int)( *(h_edgeIndex + 2*i + j)) << std::endl;
	                ( *(h_edgeIndex + i)) = (float)( *(arr + i));
	}*/

	//std::cout << "edge index matrix after sort:\n";
	//printDenseMatrix(h_edgeIndex, 2, edgeIndexSize);
	//std::cout << std::endl;
	

	return 0;
}

int getEdgeIndexSizeFromFile(const char* fileName) {

	std::ifstream dsFile(fileName);
	std::string line;

	int numOfEdges = 0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			numOfEdges++;
		}
	}
	else {
		std::cout << "Could not open the edgeIndex dataset file!\n";
		return -1;
	}

	return numOfEdges;

}

void loadEdgeIndexFromFile(const char* fileName, float* edgeIndex, const int numOfEdges,
			   std::unordered_map<int, int> &nodeMap) {

	std::ifstream dsFile(fileName);
	std::string line;

	int i=0, j=0;

	if(dsFile.is_open()) {
		while(getline(dsFile, line)) {
			std::istringstream ss(line);
			std::string word;
			while(std::getline(ss, word, '\t')) {
				*(edgeIndex + (numOfEdges*i) + j) = nodeMap.find(std::stof(word))->second;
				i = 1;
			}
		j++;
		i = 0;
		}
		std::cout << "DEBUG: edgeIndex loaded! j=" << j << std::endl;
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
	}

}

void loadEdgeIndexFromFile2(const char* fileName, float* edgeIndex, const int numOfEdges,
                           std::unordered_map<int, int> &nodeMap) {

        std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
                        std::istringstream ss(line);
                        std::string word;
                        while(std::getline(ss, word, '\t')) {
                                *(edgeIndex + i + j) = nodeMap.find(std::stof(word))->second;
                                i = 1;
                        }
                j++;
                i = 0;
                }
                std::cout << "DEBUG: edgeIndex loaded! j=" << j << std::endl;
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
        }

}

int getFeatureSizeFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
	std::string line, word;

	int numOfFeatures = -1;

	if(dsFile.is_open()) {
		getline(dsFile, line);
		std::istringstream ss(line);
		while(std::getline(ss, word, '\t')) {
			numOfFeatures++;
		}
	}
	else {
		std::cout << "Could not open the feature dataset file!\n";
		return -1;
	}

	return numOfFeatures;
}

int getNumOfNodesFromFile(const char* fileName) {
	std::ifstream dsFile(fileName);
        std::string line;

        int numOfNodes = 0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
			numOfNodes++;
		}
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
                return -1;
        }

        return numOfNodes;

}

void loadFeatureVectorFromFile(const char* fileName, float* featureVector, int featureSize, std::unordered_map<int, int> &nodeMap) {
	std::ifstream dsFile(fileName);
        std::string line;

        int i=0, j=0;

        if(dsFile.is_open()) {
                while(getline(dsFile, line)) {
                        std::istringstream ss(line);
                        std::string word;
			std::getline(ss, word, '\t'); // escape node index
			nodeMap.insert({std::stof(word),i});
                        while(std::getline(ss, word, '\t')) {
				if(word.length() < 5) {
                                    *(featureVector + i*featureSize + j) = std::stof(word);
                                    j += 1;
				}
                        }
                i++;
                j = 0;
                }
        }
        else {
                std::cout << "Could not open the feature dataset file!\n";
        }
}


#ifdef __cplusplus
}
#endif
